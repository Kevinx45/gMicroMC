#include "hip/hip_runtime.h"
#ifndef __REALTIME__
#define __REALTIME__
#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include "microMC_chem.h"

#define REALTIME_FILEIN "./Input/electron_broadspectrum_2022_09_19/"
#define REALTIME_FILEOUT "/home/satzhan/repos/gMicroMC/chem_stage/Results/electron_broadspectrum_2022_09_19_v20/totalrecordMETAtot_a"

// #define REALTIME_FILEIN "./Input/data80proton/"
// #define REALTIME_FILEOUT "/home/satzhan/repos/gMicroMC/chem_stage/Results/data80proton_v13_us_a/totalrecordMETAtot"
#define FILEOH "/OH_1ns"
#define FILEOHNAME "1ns"

void printDevProp(int device)
//      print out device properties
{
    int devCount;
    hipDeviceProp_t devProp;
//      device properties

    hipGetDeviceCount(&devCount);
	cout << "Number of device:              " << devCount << endl;
	cout << "Using device #:                " << device << endl;
    hipGetDeviceProperties(&devProp, device);
	
	printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %7.2f MB\n",  
	devProp.totalGlobalMem/1024.0/1024.0);
    printf("Total shared memory per block: %5.2f kB\n",  
	devProp.sharedMemPerBlock/1024.0);
    printf("Total registers per block:     %u\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    	
	printf("Maximum dimension of block:    %d*%d*%d\n", 			
	devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
	printf("Maximum dimension of grid:     %d*%d*%d\n", 
	devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
    printf("Clock rate:                    %4.2f GHz\n",  devProp.clockRate/1000000.0);
    printf("Total constant memory:         %5.2f kB\n",  devProp.totalConstMem/1024.0);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
//      obtain computing resource

}

void calDNAreact_radius(float* rDNA,float deltat)
{
	float k[5]={6.1,9.2,6.4,6.1,1.8};
	float tmp=sqrtf(PI*DiffusionOfOH*deltat*0.001);
	for(int i=0;i<5;i++)
	{
		rDNA[i]=k[i]/(4*PI*DiffusionOfOH)*10/6.023;//k 10^9 L/(mol*s), Diffusion 10^9 nm^2/s. t ps
		rDNA[i]=sqrtf(rDNA[i]*tmp+tmp*tmp*0.25)-tmp*0.5;
	}
	rDNA[5]=0;//histone protein absorption radius, assumed!!!
}

__device__ float caldistance(float3 pos1, float3 pos2)
{
	return (sqrtf((pos1.x -pos2.x)*(pos1.x -pos2.x)+(pos1.y -pos2.y)*(pos1.y -pos2.y)+(pos1.z -pos2.z)*(pos1.z -pos2.z)));
}
// accessors and mutators functions 
// can be names like variables
// is this accessor? or mutator?
// this function looks like it does something more? 
__device__ float3 PosToWall(int type, float3 pos, int index) 
{
	// xy xz yz
	float shiftz;
	float shifty;
	float shiftx;
	
	if (0 <= index && index < 4) { // xy
		// shift by -z
		shiftz = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shiftx = 0.0;
			shifty = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shiftx = 5.5/2.0;
			shifty = 0;
		}	
		if (index % 4 == 2) {
			shiftx = 0.0;
			shifty = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shiftx = -5.5/2.0;
			shifty = 0;
		}	
	}
	if (4 <= index && index < 8) { // xz
		// shift by -y
		shifty = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shiftx = 0.0;
			shiftz = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shiftx = 5.5/2.0;
			shiftz = 0;
		}	
		if (index % 4 == 2) {
			shiftx = 0.0;
			shiftz = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shiftx = -5.5/2.0;
			shiftz = 0;
		}	
	}
	if (8 <= index && index < 12) { // yz
		// shift by -x
		shiftx = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shifty = 0.0;
			shiftz = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shifty = 5.5/2.0;
			shiftz = 0;
		}	
		if (index % 4 == 2) {
			shifty = 0.0;
			shiftz = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shifty = -5.5/2.0;
			shiftz = 0;
		}	
	}
	// shift = segment center point
	// this is to shift radical position to the
	// center of the segment
	// imagine they are close to each other
	// then we need to substruct to center radical 
	// within the segment
	pos.x = pos.x - shiftx; // relative to its center ?
	pos.y = pos.y - shifty; // 
	pos.z = pos.z - shiftz;
	float xc, yc, zc; // rotate
	switch(type)
	{
		//Straight type
	case 1:////!!!!!the following needs to be revised and confirmed
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 2://-z
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;	
		break;}
	case 3://+y
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 4:
		{xc = pos.x;
		yc = pos.z;
		zc = -pos.y;
		break;}
	case 5://+x
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;
		break;}
	case 6:
		{xc = pos.z;
		yc = pos.y;
		zc = -pos.x;
		break;}
	}
	pos.x=xc;
	pos.y=yc;
	pos.z=zc;//*/
	return pos;
}

#if RANDGEO==0
__device__ float3 pos2local(int type, float3 pos, int index)
{
//do the coordinate transformation, index is the linear index for the referred box
//from global XYZ to local XYZ so that we can use the position of DNA base in two basic type (Straight and Bend) 
	int i = index%NUCLEUS_DIM;//the x,y,z index of the box
	int j = floorf((index%(NUCLEUS_DIM*NUCLEUS_DIM))/NUCLEUS_DIM);
	int k = floorf(index/NUCLEUS_DIM/NUCLEUS_DIM);
	//printf("relative to type %d %d %d %d\n", type, x,y,z);
	// this pos is the electron position which we push into the voxel
	// say N = 67
	// x is the box index [0, 67] ... 
	// here the center of the cylinder is at zero that means it can have negative coordinates
	// but x y z can't be negative as they are just box coordantes?
	// no lol
	// box is centered around zero that's okay
	// now need to push electron into it
	// x, y, z box index that we shift into global coordinate
	// 2 * x + 1 - N = [0 N] * 2 - N -> [-N N] / 2 -> [-N / 2 ; N / 2] * UL => xvec 
	// float shiftx = (2*i + 1 - NUCLEUS_DIM)*UNITLENGTH*0.5; 
	// float shifty = (2*j + 1 - NUCLEUS_DIM)*UNITLENGTH*0.5; 
	float shiftz = (k - (NUCLEUS_DIM_Z / 2)) * UNITLENGTH + UNITLENGTH * 0.5; 
	float shifty = (j - (NUCLEUS_DIM / 2)) * UNITLENGTH + UNITLENGTH * 0.5; 
	float shiftx = (i - (NUCLEUS_DIM / 2)) * UNITLENGTH + UNITLENGTH * 0.5; 
	pos.x = pos.x - shiftx; //relative to its center ?
	pos.y = pos.y - shifty; // 
	// pos.z = pos.z-(2*z + 1 - NUCLEUS_DIM_Z)*UNITLENGTH*0.5;
	pos.z = pos.z - shiftz;
	float xc, yc, zc;
	switch(type)
	{
		//Straight type
	case 1:////!!!!!the following needs to be revised and confirmed
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 2://-z
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;	
		break;}
	case 3://+y
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 4:
		{xc = pos.x;
		yc = pos.z;
		zc = -pos.y;
		break;}
	case 5://+x
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;
		break;}
	case 6:
		{xc = pos.z;
		yc = pos.y;
		zc = -pos.x;
		break;}
	case 7://Bend
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 8:
		{xc = -pos.z;//Rz(pi)Ry(pi/2) [-Ry(pi/2)] 
		yc = -pos.y;
		zc = -pos.x;
		break;}
	case 9:
		{xc = -pos.x;//Rz(pi)
		yc = -pos.y;
		zc = pos.z;
		break;}
	case 10:
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;	
		break;}
	case 11:
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;
		break;}
	case 12:
		{xc = pos.z;//Rz(pi)Ry(-pi/2)
		yc = -pos.y;
		zc = pos.x;
		break;}
	case 13:
		{xc = pos.x;//Rx(pi)
		yc = -pos.y;
		zc = -pos.z;
		break;}
	case 14:
		{xc = pos.z;//Ry(pi/2)
		yc = pos.y;
		zc = -pos.x;
		break;}
	case 15:
		{xc = pos.y;//Rz(-pi/2)
		yc = -pos.x;
		zc = pos.z;
		break;}
	case 16:
		{xc = -pos.z;//Ry(-pi/2)Rz(pi/2) +
		yc = pos.x;
		zc = -pos.y;
		break;}
	case 17:
		{xc = -pos.y;//Rz(pi/2)
		yc = pos.x;
		zc = pos.z;
		break;}
	case 18:
		{xc = -pos.z;//Rz(-pi/2)Rx(pi/2)
		yc = -pos.x;
		zc = pos.y;
		break;}
	case 19:
		{xc = pos.y;//Rz(-pi/2)Ry(pi)
		yc = pos.x;
		zc = -pos.z;
		break;}
	case 20:
		{xc = pos.z;//Rz(-pi/2)Rx(-pi/2)
		yc = -pos.x;
		zc = pos.y;
		break;}
	case 21:
		{xc = -pos.y;//Rz(pi/2)Ry(pi)
		yc = -pos.x;
		zc = -pos.z;
		break;}
	case 22:
		{xc = pos.z;//Rz(pi/2)Rx(pi/2) ??
		yc = pos.x;
		zc = pos.y;
		// -y -z +x
		break;}
	case 23:
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 24:
		{xc = -pos.y;//Rz(pi/2)Ry(pi/2)
		yc = pos.z;
		zc = -pos.x;
		break;}
	case 25:
		{xc = -pos.x;//Rx(pi/2)Ry(pi) ??
		yc = pos.z;
		zc = pos.y;
		// xzz
		break;}
	case 26:
		{xc = -pos.y;//Rx(pi/2)Rz(pi/2)
		yc = -pos.z;
		zc = pos.x;
		break;}
	case 27:
		{
		xc = pos.x;//Rx(-pi/2)
		yc =pos.z;
		zc = -pos.y;	
		break;}
	case 28:
		{xc = pos.y;//Rx(pi/2)Rz(-pi/2)
		yc = -pos.z;
		zc = -pos.x;
		// -z -x y
		break;}
	case 29:
		{xc = -pos.x;//Rx(-pi/2)Ry(pi) ?
		yc = -pos.z;
		zc = -pos.y;
		break;}
	case 30:
		{xc = pos.y;//Rz(-pi/2)Ry(-pi/2)
		yc = pos.z;
		zc = pos.x;
		break;}
	default:
	    {printf("wrong type %d\n", type);  // for test
		break;}
	}
	pos.x=xc;
	pos.y=yc;
	pos.z=zc;//*/
	return pos;
}

__device__ float dist_function_sqr(float3 &a, float3 &b, float &height_up, float &height_down) {
	// two coordinates a and b
	// a = event
	// b = X-chromosome
	// if dist in x direction is radius within the dist then ok
	// if dist in y direction is diameter + 50 distance then ok
	// if dist in z direction is event - chrom (if event is lower a-b < 0 height down)
	// height down is negative
	return (abs(a.x - b.x) <= CYLINDERRADIUS) 
		&& (abs(a.y - b.y) <= 50 + CYLINDERRADIUS * 2)
		&& (height_down <= a.z - b.z) 
		&& (a.z - b.z <= height_up);
}
__device__ bool withinCylinder(float3 &a, float3 &cylinder) {
	// check height
	// a = 7204.624023 -229.854507 14569.363281 :: 7171.703613 -365.000000 14413.875977
	if ((a.z < (cylinder.z - CYLINDERHEIGHT / 2)) || 
	    ((cylinder.z + CYLINDERHEIGHT / 2) < a.z)) {
		return 0;
	}
	// check radial distance
	if ((a.x - cylinder.x) * (a.x - cylinder.x) + 
		(a.y - cylinder.y) * (a.y - cylinder.y) > 
		CYLINDERRADIUS * CYLINDERRADIUS) {
		return 0;
	}
	return 1;
}
__global__ void chemSearch(
	int num, 
	Edeposit* d_edrop, 
	int* dev_chromatinIndex,
	int* dev_chromatinStart,
	int* dev_chromatinType, 
	CoorBasePair* dev_straightChrom, 
	CoorBasePair* dev_segmentChrom,
	CoorBasePair* dev_bendChrom,
	float3* dev_straightHistone,
	float3* dev_bendHistone, 
	combinePhysics* d_recorde,
	float3 *dev_chromosome, 
	int *dev_chromosome_type,
	int *dev_segmentIndex, 
	int *dev_segmentStart, 
	int *dev_segmentType)
{
	int id = blockIdx.x*blockDim.x+ threadIdx.x;
	hiprandState localState = cuseed[id%MAXNUMPAR2];
	float3 newpos, pos_cur_target;
	int3 index;
	CoorBasePair* chrom;
	float3 *histone;
	int chromNum, histoneNum,flag=0;
	while(id<num)
	{
		d_recorde[id].site.x=-1;//initialize
		d_recorde[id].site.y=-1;
		d_recorde[id].site.z=-1;
		d_recorde[id].site.w=-1;		
		d_recorde[id].prob1 = 1; //hiprand_uniform(&localState); // 1
		// d_recorde[id].prob1=1;
		d_recorde[id].prob2 = 0.0; // 0.6 ? 
		// threshold for prob2 ?
		
		pos_cur_target=d_edrop[id].position; // electron position / event position
		
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// here we need to modify cur position based on dev_chromosome and dev_chromosome_type
		// Step 0) skip events too far from the y=0 plane
		// if (abs(pos_cur_target.y) > 50 + CYLINDERRADIUS * 2) {
		// 	// y position is too far from the center
		// 	id+=blockDim.x*gridDim.x;
		// 	continue ;
		// }

		// Step 1) Find nearest chromosome :)
		int found_nearest_chromosome = 0;
		int id_chromosome = -1;
		for (int i = NUMCHROMOSOMES - 1; i >= 0 ; i--) { // 46
			float height_up = (dev_chromosome_type[i] / 2) * CYLINDERHEIGHT + CYLINDERHEIGHT / 2;
			float height_down = -(((dev_chromosome_type[i] - 1) / 2) * CYLINDERHEIGHT + CYLINDERHEIGHT / 2);
			// height_down is negative
			// type / 2 + 1 ~ 10 / 2 + 1 = 5
			if (dist_function_sqr(pos_cur_target, dev_chromosome[i], height_up, height_down)) {
				// FOUND NEAREST CHROMOSOME!
				// Step 1.1) Mark
				found_nearest_chromosome = 1;
				id_chromosome = i;
				break;
			}
		}
		
		if (found_nearest_chromosome == 0) {
			// Step 1.2) if for this radical we did not find
			// anything nearby, then continue to the next :) 
			id+=blockDim.x*gridDim.x;
			continue ;
		}
		
		// if we are here means we found chromosome
		// Step 1.3) Find nearest cylinder!
		int ttype = dev_chromosome_type[id_chromosome];
		int upper_part = ttype / 2;  // typy 4 :: 4 / 2 = 2 || type 5 :: 5 / 2 = 2
		int lower_part = (ttype - 1) / 2; // type 4 :: 4 / 2 - 1 = 1 || type 5 :: 5 / 2 - 1 = 1
		float3 nearest = dev_chromosome[id_chromosome];
		int found_cylinder = 0;
		int id_cylinder = -1;
		// check lower and upper parts, cylinders
		// now we need to redo cylinder ID, to be in range [1 10] instead of [0 9]
		for (int idy = 0; idy < upper_part; idy++) {
			// Step 1.4) Check left and right cylinders 
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - 50 - CYLINDERRADIUS; 
			left_shift.z = nearest.z + CYLINDERHEIGHT * (idy + 1);
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + 50 + CYLINDERRADIUS; 
			right_shift.z = nearest.z + CYLINDERHEIGHT * (idy + 1);
			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome;
				id_cylinder = idy + lower_part + 1;
				break;
			}
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
				id_cylinder = idy + lower_part + 1;
				break;
			}
		}
		// 
		if (!found_cylinder) {
			for (int idy = 0; idy < lower_part; idy++) {
				// Step 1.4) Check left and right cylinders 
				float3 left_shift;
				left_shift.x = nearest.x + 0.0;
				left_shift.y = nearest.y - 50 - CYLINDERRADIUS; 
				left_shift.z = nearest.z - CYLINDERHEIGHT * (idy + 1); // 
				float3 right_shift;
				right_shift.x = nearest.x + 0.0;
				right_shift.y = nearest.y + 50 + CYLINDERRADIUS; 
				right_shift.z = nearest.z - CYLINDERHEIGHT * (idy + 1);
				if (withinCylinder(pos_cur_target, left_shift)) {
					pos_cur_target.x -= left_shift.x;
					pos_cur_target.y -= left_shift.y;
					pos_cur_target.z -= left_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome;
					id_cylinder = lower_part - 1 - idy;
					break;
				}
				if (withinCylinder(pos_cur_target, right_shift)) {
					pos_cur_target.x -= right_shift.x;
					pos_cur_target.y -= right_shift.y;
					pos_cur_target.z -= right_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
					id_cylinder = lower_part - 1 - idy;
					break;
				}		
			}
		}
		// Step 1.5) check middle part
		if (!found_cylinder) {
			// printf("Middle check\n");
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - CYLINDERRADIUS; 
			left_shift.z = nearest.z + 0.0;
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + CYLINDERRADIUS; 
			right_shift.z = nearest.z + 0.0;

			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome; // left side chromosome
				id_cylinder = lower_part;
			}
			else
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
				id_cylinder = lower_part;
			}
		}
		// if (id < 10) {
		// 	printf("Chromosome id type :: %d %d\n", dev_chromosome_type[id_chromosome], found_cylinder);
		// }	

		if (!found_cylinder) {
			id+=blockDim.x*gridDim.x;
			continue ;
		}
		// cylinder was found and shifted appropiately
		// continue as usual
		// END OF STEP 1
		// *******************************************
		// *******************************************
		// *******************************************
		// *******************************************
		
		// from the global coordinate (-min max) to [0 N] index coordinate
		// what we know is that z must be say 6
		index.x=floorf(pos_cur_target.x/UNITLENGTH) + (NUCLEUS_DIM/2); // 2000 
		index.y=floorf(pos_cur_target.y/UNITLENGTH) + (NUCLEUS_DIM/2);
		index.z=floorf(pos_cur_target.z/UNITLENGTH) + (NUCLEUS_DIM_Z/2);
		
		// printf("It thinks Nucleosome index is %d %d %d\n", 
		// 	index.x, index.y, index.z
		// );

		int delta=index.x+index.y*NUCLEUS_DIM+index.z*NUCLEUS_DIM*NUCLEUS_DIM,minindex=-1;
		float distance[3]={100},mindis=100;
		// TO DO
		// just check 1 extra voxel nearby the wall
		flag=0;
		
		// flag changed range from 0-27 to 13-14
		for(int i=0;i<27;i++) // +6 walls
		{
			int newindex = delta+neighborindex[i];
			// if (i == 13) {
			// 	printf("ID check %d %d\n", newindex, delta);
			// }

			// flag changed Z
			if(newindex<0 || newindex > NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z-1) continue;
			int type = dev_chromatinType[newindex];
			// if (i == 13) { 
			// 	printf("Type check %d\n", type);
			// }
			if(type==-1 || type==0) continue;

			newpos = pos2local(type, pos_cur_target, newindex);
			if(type<7)
			{
				chrom=dev_straightChrom;
				chromNum=STRAIGHT_BP_NUM;
				histone=dev_straightHistone;
				histoneNum=STRAIGHT_HISTONE_NUM;
			}
			else
			{
				chrom=dev_bendChrom;
				chromNum=BEND_BP_NUM;
				histone=dev_bendHistone;
				histoneNum=BEND_HISTONE_NUM;
			}
			if(flag) break;
			for(int j=0;j<chromNum;j++) // 200 nucleosome
			{
				// can take the size of base into consideration, distance should be distance-r;
				mindis=100,minindex=-1;
				distance[0] = caldistance(newpos, chrom[j].base)-RBASE;
				distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR;
				distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR;
				for(int iii=0;iii<3;iii++)
				{
					if(mindis>distance[iii])
					{
						mindis=distance[iii];
						minindex=iii;
					}
				}
				if(mindis<0)
				{
					if(minindex>0)
					{
						// GEANT4  
						d_recorde[id].site.x = id_chromosome; // 
						d_recorde[id].site.y = (dev_chromatinStart[newindex]+j) + TOTALBP * id_cylinder;  
						d_recorde[id].site.z = 3+minindex;
						d_recorde[id].site.w = 1; // phys or chem 0/1
					}
					flag=1;
					break;
				}
				int tmp = floorf(hiprand_uniform(&localState)/0.25);
				distance[0] = caldistance(newpos, chrom[j].base)-RBASE-d_rDNA[tmp];
				distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR- d_rDNA[4];
				distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR- d_rDNA[4];
				for(int iii=0;iii<3;iii++)
				{
					if(mindis>distance[iii])
					{
						mindis=distance[iii];
						minindex=iii;
					}
				}	
				if(mindis<0)
				{
					if(minindex>0)
					{
						// event thread id 
						d_recorde[id].site.x = id_chromosome; 
						d_recorde[id].site.y = (dev_chromatinStart[newindex]+j) + TOTALBP * id_cylinder;  
						// X-chromosome id //  
						d_recorde[id].site.z = 3+minindex; // left or right
						d_recorde[id].site.w = 1; // chem
					}
					flag=1;
					break;
				}
			}
			if(flag) break;
		}
		// Do all 6 walls * 4 each
		// CURRENT UPDATE 05/20/2022 **************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		if (flag == 0) { // still not found

			for(int i = 0; i < 24 && flag == 0; i++) // +6 walls
			{
				// printf("Checking segment number %d\n", i);
				int newdelta = delta;
				// conversions
				// the first 12 are in the current voxel so we don't need to change delta
				// 
				if (i >= 12) { // xy xz yz
					if (i < 16)  // xy +1z
						newdelta = delta + NUCLEUS_DIM*NUCLEUS_DIM;
					else if (i < 20) // xz +1y
						newdelta = delta + NUCLEUS_DIM;
					else // yz +1x
						newdelta = delta + 1;
				}
				int newindex = newdelta * 12 + i % 12;

				// ************flag changed Z
				// printf("New index vs total volume :: %d vs %d\n", newindex, NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z * 12);
				if(newindex<0 || newindex >= NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z * 12) continue;
				
				int type = dev_segmentType[newindex];
				if(type==-1 || type==0) continue;

				// type is not used for pos2local because we are just getting next to this cell
				// no rotation is needed here
				float3 pos_within_voxel = pos2local(1, pos_cur_target, newdelta);
				// the idea here is to shift first the position within the voxel?
				// shift relative to the voxel
				// then choose to shift next to wall center?
				// 
				newpos = PosToWall(type, pos_within_voxel, i % 12);
				// printf("Id %d and relative poistion: %0.2f %0.2f %0.2f\n", i, newpos.x, newpos.y, newpos.z);
				if(type<7)
				{
					chrom=dev_segmentChrom;
					chromNum=SEGMENT_BP_NUM;
				}
				else {
					// it's an error :)
				}
				if(flag) break;
				for(int j=0;j<chromNum;j++) // 17 SEGMENT
				{
					// can take the size of base into consideration, distance should be distance-r;
					mindis=100,minindex=-1;
					distance[0] = caldistance(newpos, chrom[j].base)-RBASE;
					distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR;
					distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR;
					
					for(int iii=0;iii<3;iii++)
					{
						if(mindis>distance[iii])
						{
							mindis=distance[iii];
							minindex=iii;
						}
					}
					if(mindis<0)
					{
						if(minindex>0)
						{
							// id is correct in the sense that it belongs to the 
							// event radical id
							// so we can record here anything
							// but what we need is the Chromosome ID to distinguish different DNA
							// base pair ID for damage calculations
							// and right or left dmg pair.
							d_recorde[id].site.x = id_chromosome; 
							d_recorde[id].site.y = (dev_segmentStart[newindex]+j) + TOTALBP * id_cylinder;  
							d_recorde[id].site.z = 3+minindex;
							d_recorde[id].site.w = 1; // phys or chem
						}
						flag=1; // found
						break;
					}
					int tmp = floorf(hiprand_uniform(&localState)/0.25);
					distance[0] = caldistance(newpos, chrom[j].base)-RBASE-d_rDNA[tmp];
					distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR- d_rDNA[4];
					distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR- d_rDNA[4];
					for(int iii=0;iii<3;iii++)
					{
						if(mindis>distance[iii])
						{
							mindis=distance[iii];
							minindex=iii;
						}
					}	
					if(mindis<0)
					{
						if(minindex>0)
						{
							d_recorde[id].site.x = id_chromosome; 
							d_recorde[id].site.y = (dev_segmentStart[newindex]+j) + TOTALBP * id_cylinder;  
							d_recorde[id].site.z = 3+minindex;
							d_recorde[id].site.w = 1; // phys or chem
						}
						flag=1;
						break;
					}
				}
				if(flag) break;
			}
		}
		id+=blockDim.x*gridDim.x;
	}
	cuseed[id%MAXNUMPAR2]=localState;
}

__global__ void phySearch(
	int num, 
	Edeposit* d_edrop, 
	int* dev_chromatinIndex,
	int* dev_chromatinStart,
	int* dev_chromatinType, 
	CoorBasePair* dev_straightChrom,
	CoorBasePair* dev_segmentChrom,
	CoorBasePair* dev_bendChrom,
	float3* dev_straightHistone,
	float3* dev_bendHistone, 
	combinePhysics* d_recorde,
	float3 *dev_chromosome, 
	int *dev_chromosome_type,
	int *dev_segmentIndex,
	int *dev_segmentStart, 
	int *dev_segmentType)
{
	int id = blockIdx.x*blockDim.x+ threadIdx.x;
	hiprandState localState = cuseed[id%MAXNUMPAR2];
	float3 newpos, pos_cur_target;
	int3 index;
	CoorBasePair* chrom;
	float3 *histone;
	int chromNum, histoneNum,flag=0;
	while(id<num)
	{
		d_recorde[id].site.x=-1;//initialize
		d_recorde[id].site.y=-1;
		d_recorde[id].site.z=-1;
		d_recorde[id].site.w=-1;		
		d_recorde[id].prob1=d_edrop[id].e;
		d_recorde[id].prob2=0.0; //hiprand_uniform(&localState)*(EMAX-EMIN) + EMIN; // constant 
		// threshold for prob2

		pos_cur_target=d_edrop[id].position;
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// here we need to modify cur position based on dev_chromosome and dev_chromosome_type
		// Step 0) skip events too far from the y=0 plane
		// if (id < num) {
		// 	printf("Physical search %d %d\n", id, num);
		// }

		// if (abs(pos_cur_target.y) > 50 + CYLINDERRADIUS * 2) {
		// 	// y position is too far from the center
		// 	id+=blockDim.x*gridDim.x;
		// 	continue ;
		// }
		// printf("Within the plane! %d\n", id);
		// Step 1) Find nearest chromosome :)
		bool found_nearest_chromosome = 0;
		int id_chromosome = -1;
		for (int i = 0; i < NUMCHROMOSOMES; i++) {
			float height_up = (dev_chromosome_type[i] / 2) * CYLINDERHEIGHT + CYLINDERHEIGHT / 2;
			float height_down = -(((dev_chromosome_type[i] - 1) / 2) * CYLINDERHEIGHT + CYLINDERHEIGHT / 2);
			if (dist_function_sqr(pos_cur_target, dev_chromosome[i], height_up, height_down)) {
				// FOUND NEAREST CHROMOSOME!
				// Step 1.1) Mark
				found_nearest_chromosome = 1;
				id_chromosome = i;
				break;
			}
		}
		if (found_nearest_chromosome == 0) {
			// Step 1.2) if for this radical we did not find
			// anything nearby, then continue to the next :) 
			id+=blockDim.x*gridDim.x;
			continue ;
		}
		// printf("Near some Chromosome! %d %d\n", id, id_chromosome);
		// if we are here means we found chromosome
		// Step 1.3) Find nearest cylinder!
		int ttype = dev_chromosome_type[id_chromosome];
		int upper_part = ttype / 2;  // typy 4 :: 4 / 2 = 2 || type 5 :: 5 / 2 = 2
		int lower_part = (ttype - 1) / 2; // type 4 :: (4 - 1) / 2 = 1 || type 5 :: (5 - 1) / 2 = 2
		float3 nearest = dev_chromosome[id_chromosome];
		// printf("Cur pos and chromosome %f %f %f && %f %f %f\n", 
		// 	pos_cur_target.x, pos_cur_target.y, pos_cur_target.z, 
		// 	nearest.x, nearest.y, nearest.z);
		bool found_cylinder = 0;
		int id_cylinder = -1;
		// check lower and upper parts, cylinders
		for (int idy = 0; idy < upper_part; idy++) {
			// Step 1.4) Check left and right cylinders 
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - 50 - CYLINDERRADIUS; 
			left_shift.z = nearest.z + CYLINDERHEIGHT * (idy + 1);
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + 50 + CYLINDERRADIUS; 
			right_shift.z = nearest.z + CYLINDERHEIGHT * (idy + 1);
			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome;
				id_cylinder = idy + lower_part + 1;
				break;
			}
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
				id_cylinder = idy + lower_part + 1;
				break;
			}
		}
		// 
		if (!found_cylinder) {
			for (int idy = 0; idy < lower_part; idy++) {
				// Step 1.4) Check left and right cylinders 
				float3 left_shift;
				left_shift.x = nearest.x + 0.0;
				left_shift.y = nearest.y - 50 - CYLINDERRADIUS; 
				left_shift.z = nearest.z - CYLINDERHEIGHT * (idy + 1);
				float3 right_shift;
				right_shift.x = nearest.x + 0.0;
				right_shift.y = nearest.y + 50 + CYLINDERRADIUS; 
				right_shift.z = nearest.z - CYLINDERHEIGHT * (idy + 1);
				if (withinCylinder(pos_cur_target, left_shift)) {
					pos_cur_target.x -= left_shift.x;
					pos_cur_target.y -= left_shift.y;
					pos_cur_target.z -= left_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome;
					id_cylinder = lower_part - 1 - idy;
					break;
				}
				if (withinCylinder(pos_cur_target, right_shift)) {
					pos_cur_target.x -= right_shift.x;
					pos_cur_target.y -= right_shift.y;
					pos_cur_target.z -= right_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
					id_cylinder = lower_part - 1 - idy;
					break;
				}		
			}
		}
		// Step 1.5) check middle part
		if (!found_cylinder) {
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - CYLINDERRADIUS; 
			left_shift.z = nearest.z + 0.0;
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + CYLINDERRADIUS; 
			right_shift.z = nearest.z + 0.0;
			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome; // left side chromosome
				id_cylinder = lower_part;
			}
			else
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES; // right side chromosome
				id_cylinder = lower_part;
			}
		}

		if (!found_cylinder) {
			id+=blockDim.x*gridDim.x; // event id damage deposition id
			continue ;
		}
		// printf("Within some cylinder! %d\n", id);
		// cylinder was found and shifted appropiately
		// continue as usual
		// END OF STEP 1
		// *******************************************
		// *******************************************
		// *******************************************
		// *******************************************
		
		index.x=floorf(pos_cur_target.x/UNITLENGTH) + (NUCLEUS_DIM/2); // 2000 
		index.y=floorf(pos_cur_target.y/UNITLENGTH) + (NUCLEUS_DIM/2);
		index.z=floorf(pos_cur_target.z/UNITLENGTH) + (NUCLEUS_DIM_Z/2);
		// printf("It thinks Nucleosome index is %d %d %d\n", 
		// 	index.x, index.y, index.z
		// );
		int delta=index.x+index.y*NUCLEUS_DIM+index.z*NUCLEUS_DIM*NUCLEUS_DIM,minindex=-1;
		float distance[3]={100},mindis=100;
		for(int i=0;i<27;i++)
		{
			flag=0;
			int newindex = delta+neighborindex[i];
			if(newindex<0 || newindex > NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z-1) continue;
			int type = dev_chromatinType[newindex];
			if(type==-1 || type==0) continue;

			newpos = pos2local(type, pos_cur_target, newindex);
			// if (id < 10) {
				// printf("type = %d\n", type);
				// printf("local pos %f %f %f\n", newpos.x, newpos.y, newpos.z);
			// }
			if(type<7)
			{
				chrom=dev_straightChrom;
				chromNum=STRAIGHT_BP_NUM;
				histone=dev_straightHistone;
				histoneNum=STRAIGHT_HISTONE_NUM;
			}
			else
			{
				chrom=dev_bendChrom;
				chromNum=BEND_BP_NUM;
				histone=dev_bendHistone;
				histoneNum=BEND_HISTONE_NUM;
			}
			// for(int j=0;j<histoneNum;j++)
			// {
			// 	mindis = caldistance(newpos, histone[j]) - RHISTONE;
			// 	if(mindis < 0) flag=1;
			// }
			// printf("flag lol %d\n", flag);
			if(flag) break;
			// printf("Avoided flag\n");
			for(int j=0;j<chromNum;j++) // 200 // nucleosome
			{
				// can take the size of base into consideration, distance should be distance-r;
				mindis=100,minindex=-1;
				distance[0] = caldistance(newpos, chrom[j].base)-RBASE-RPHYS;
				distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR- RPHYS;
				distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR- RPHYS;
				for(int iii=0;iii<3;iii++)
				{
					if(mindis>distance[iii])
					{
						mindis=distance[iii];
						minindex=iii;
					}
				}
				// 
				// if (mindis < 1.0) {
				// 	printf("mindis %f  and bp_id = %d\n", mindis, j);
				// }
				if(mindis<0)
				{
					// printf("found mindis %f\n", mindis);
					if(minindex>0)
					{
						//[10 8 7 6 6 .... ] 
						// [0 1 2 3 4] index of a X-chromosome
						// [10 19 26 ... ] cylinder index
						// 120000 * 200 * 522 cylinders 12,000,000,000
						// we don't need site.x previous definition
						// we do need to add cylinder ID and chromosome
						// printf("found x\n");
						// 26813034
						d_recorde[id].site.x = id_chromosome; 
						d_recorde[id].site.y = (dev_chromatinStart[newindex]+j) + TOTALBP * id_cylinder;  
						d_recorde[id].site.z = 3+minindex;
						d_recorde[id].site.w = 0;
						if (d_recorde[id].site.y == 8290192) {
							printf("Voxel Found!\n");
							printf("Related chromosome type :: %d\n", dev_chromosome_type[id_chromosome]);
							printf("Related chromosome ID :: %d\n", id_chromosome);
							printf("Related cylinder ID :: %d\n", id_cylinder);
							printf("Related index :: %d %d %d\n", index.x, index.y, index.z);
							printf("voxel related pos original :: %f %f %f\n", 
								d_edrop[id].position.x, 
								d_edrop[id].position.y, 
								d_edrop[id].position.z);
							printf("voxel related phy energy :: %f", d_edrop[id].e);

						}
					}
					flag=1;
				}
			}
			if(flag) break;
		}
		// Do all 6 walls * 4 each
		// CURRENT UPDATE 05/20/2022 **************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		if (flag == 0) { // still not found

			for(int i = 0; i < 24 && flag == 0; i++) // +6 walls
			{
				// printf("Checking segment number %d\n", i);
				int newdelta = delta;
				// conversions
				// the first 12 are in the current voxel so we don't need to change delta
				// 
				if (i >= 12) { // xy xz yz
					if (i < 16)  // xy +1z
						newdelta = delta + NUCLEUS_DIM*NUCLEUS_DIM;
					else if (i < 20) // xz +1y
						newdelta = delta + NUCLEUS_DIM;
					else // yz +1x
						newdelta = delta + 1;
				}
				int newindex = newdelta * 12 + i % 12;

				// ************flag changed Z
				// printf("New index vs total volume :: %d vs %d\n", newindex, NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z * 12);
				if(newindex<0 || newindex >= NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z * 12) continue;
				
				int type = dev_segmentType[newindex];
				if(type==-1 || type==0) continue;

				float3 pos_within_voxel = pos2local(1, pos_cur_target, newdelta);
				newpos = PosToWall(type, pos_within_voxel, i % 12);
				if(type<7)
				{
					chrom=dev_segmentChrom;
					chromNum=SEGMENT_BP_NUM;
				}
				else {
					// it's an error :)
				}
				if(flag) break;
				for(int j=0;j<chromNum;j++) // 17 SEGMENT
				{
					// can take the size of base into consideration, distance should be distance-r;
					mindis=100,minindex=-1;
					distance[0] = caldistance(newpos, chrom[j].base) - RBASE - RPHYS;
					distance[1] = caldistance(newpos, chrom[j].left) - RSUGAR - RPHYS;
					distance[2] = caldistance(newpos, chrom[j].right) - RSUGAR - RPHYS;
					
					for(int iii=0;iii<3;iii++)
					{
						if(mindis>distance[iii])
						{
							mindis=distance[iii];
							minindex=iii;
						}
					}
					if(mindis<0)
					{
						if(minindex>0)
						{
							// id is correct in the sense that it belongs to the 
							// event radical id
							// so we can record here anything
							// but what we need is the Chromosome ID to distinguish different DNA
							// base pair ID for damage calculations
							// and right or left dmg pair.
							// printf("Found something xD\n");
							d_recorde[id].site.x = id_chromosome;  // 92 ? 
							d_recorde[id].site.y = (dev_segmentStart[newindex]+j) + TOTALBP * id_cylinder;  
							d_recorde[id].site.z = 3+minindex; // left or right pair
							d_recorde[id].site.w = 0; // phys or chem
						}
						flag=1; // found
						break;
					}
				}
				if(flag) break;
			}
		}	
		//if(id%(blockDim.x*gridDim.x)==0) printf("id is %d\n", id);
		id+=blockDim.x*gridDim.x;//*/
	}
	cuseed[id%MAXNUMPAR2]=localState;
}//*/
#endif
/***********************************************************************************/

Edeposit* readStage(int *numPhy, int mode, int file_id)
/*******************************************************************
c*    Reads electron reactive events from physics stage result     *
c*    Setup electron events as a list for the DNA damages          *
output *effphy 
Number of effective Physics damage
c******************************************************************/
{
	int start,stop;
	float data[4];
	Edeposit *hs = NULL;
	int len = 0, prev_len = 0;
	{
		// cout << file_id << " ";
		ifstream infile;
		if(mode==0) {
			string input = REALTIME_FILEIN + to_string(file_id) + "/totalphy.dat"; 
			infile.open(input,ios::binary);
			// printf("physics results: Reading %s\n", input.c_str());
		}	
		else {
			string input = REALTIME_FILEIN + to_string(file_id) + FILEOH + ".dat";
			infile.open(input,ios::binary);
			// printf("chemistry results: Reading %s\n", input.c_str());
		}
		start=infile.tellg();
		infile.seekg(0, ios::end);
		stop=infile.tellg();
		len=(stop-start)/16;
		if(len==0) { infile.close(); return hs; }
		infile.seekg(0, ios::beg);
		hs = (Edeposit *)malloc(sizeof(Edeposit)*(prev_len + len));
		
		for(int j=prev_len;j<prev_len + len;j++)
		{
			infile.read(reinterpret_cast <char*> (&data), sizeof(data));
			hs[j].position.x=data[0];
			hs[j].position.y=data[1];
			hs[j].position.z=data[2];
			if(mode==0) hs[j].e=data[3];
			else hs[j].e=1-PROBCHEM;
		} 
		prev_len += len;
		infile.close();
	} 
	// cout << endl;
	(*numPhy) += prev_len;
 	return hs;
}

void quicksort(chemReact*  hits,int start, int stop, int sorttype)
{   
    //CPU sort function for ordering chemReacts in cpu memory
    switch(sorttype)
    {
	    case 1:
	    {   sort(hits+start,hits+stop,compare1);
	        break;
	    }
	    case 2:
	    {   sort(hits+start,hits+stop,compare2);
	        break;
	    }
	    default:
	    {   sort(hits+start,hits+stop,compare1);
	        break;
	    }
    }
}
chemReact* combinePhy(int* totalphy, combinePhysics* recorde, int mode, int file_id)
{
	int counts=(*totalphy);
	sort(recorde,recorde+counts,compare3);
	
	int j,num=0;
	// printf("CombinePhy counts %d\n", counts);
    for(int i=0; i<counts;)
    {
		if (recorde[i].site.z==-1) {i++;continue;}
    	j=i+1;
        while(recorde[j].site.x==recorde[i].site.x) // id base pair
        {
        	if(recorde[j].site.y==recorde[i].site.y && recorde[j].site.z==recorde[i].site.z)
        	{
        		if (mode == 0) recorde[i].prob1 +=recorde[j].prob1; // sum up energies
        		else recorde[i].prob2 *= recorde[j].prob2; // mode 1 for chem stage 
				// for mode 1: reduction of the threshold
        		recorde[j].site.z=-1;
        	}
        	j++;
        	if(j==counts) break;
        }        	
        i++;
    }
	// why are we doing these counts?
    for(int i=0;i<counts;i++)
    {
		if(recorde[i].site.z!=-1 && recorde[i].prob2<recorde[i].prob1)
    	{
    		num++;
    	}
    }
	// printf("counts after probabilities %d\n", num);
    if(num==0) {(*totalphy)=0;return NULL;}
	string output = REALTIME_FILEOUT + to_string(file_id) + "_" + FILEOHNAME + ".txt";
	cout << output << endl;

	ofstream fout;
	if (mode == 0) { // at first open and replace
		fout.open(output.c_str());
	}
	else { // then append
		fout.open(output.c_str(), std::ios_base::app);
	}
	
    chemReact* recordPhy=(chemReact*) malloc(sizeof(chemReact)*num);
    int index=0;
    for(int i=0;i<counts;i++)
    {
		if (recorde[i].site.z != -1) { // if found event near to base pair
			fout << mode << " " << // mode  //0 
				recorde[i].site.x << " " <<  // chrom id // 1
				recorde[i].site.y << " " <<  // bp id // 2
				recorde[i].site.z << " " <<  // left/right // 3
				recorde[i].site.w << " " <<  // 1/0 phy/chem // 5
				recorde[i].prob1 << endl; // energy or prob 0.6
		}
    	if(recorde[i].site.z!=-1 && recorde[i].prob2<recorde[i].prob1)
    	{
    		recordPhy[index].x=recorde[i].site.x;
    		recordPhy[index].y=recorde[i].site.y;
    		recordPhy[index].z=recorde[i].site.z;
    		recordPhy[index].w=recorde[i].site.w;
    		index++;
    	}
    }
    (*totalphy)=num;
    return recordPhy;
}
void damageAnalysis(int counts, chemReact* recordpos, int numFiles, float dose)
{
	// seems currently only the number of total SSB or DSB are correct
	// be careful to use the number in each category!!
	if(counts==0) return;
	char buffer[256];
	int complexity[7]={0};//SSB, 2xSSB, SSB+, 2SSB, DSB, DSB+, DSB++
	int results[7]={0}; //SSBd,  SSbi, SSbm, DSBd, DSBi, DSBm, DSBh.
	
	quicksort(recordpos,0,counts,1);
	// sort, this sorts x first which id DNA id, then it sorts by bp id
    int start=0,m,numofstrand,numoftype,k,cur_dsb;
	// printf("counts %d\n", counts);
    for(int i=0; i<counts;) // go over all one by one?
    {
    	if(recordpos[i].z==-1) {i++;continue;} // skip if it's not damaged? why recorded lol
    	start=i;
        while(i<counts)
        {
        	if(recordpos[i].x==recordpos[start].x) i++; // if it's the same DNA ?
        	else break;
        }
		// printf("Ids of the record and next? :: %d %d\n", start, i);
        if(i==start+1)//only one break on the DNA whole DNA, rather it's the end of the DNA ... 
        {
        	complexity[0]++; // single break
        	results[recordpos[start].w]++;
        	continue;//find breaks in another DNA
        }
		// range [start -> i] same DNA 
        if(i>start+1) quicksort(recordpos,start,i,2);//order damage sites so that search can be done ?
		cur_dsb=0;
        for(k=start;k<i-1;)//more than one break range [k -> i) of the DNA
        {
        	if(recordpos[k+1].y-recordpos[k].y>dS)
        	{
        		complexity[1]++; // 2xSSB
        		results[recordpos[k].w]++;
        		k++;
        		continue;
        	}
        	else
        	{
	        	m=k+1;
	        	numoftype=0;
	        	numofstrand=0;
	        	int flag=0;//means SSB, 1 for DSB
        		while(m<i)
        		{
        			if( recordpos[m].z!=recordpos[m-1].z)//recordpos[m].y-recordpos[m-1].y<dDSB &&
        			{ // left + right
        				numofstrand++;
        				if(recordpos[m].w!=recordpos[k].w) numoftype++; // phys+chem
        				int j=m; // next
        				int tmptype=0;
        				for(;j>k-1;j--) // go back?
        				{
        					if(recordpos[m].y-recordpos[j].y>dDSB) break; // more than 10 
        					if(recordpos[j].w!=recordpos[k].w) tmptype++; // phys+chem
        				}

        				if(j==k-1) flag=1;//DSB k->m less then 10 all
        				else if(j==k && m==k+1) flag=2;//2SSB didn't reach k-> m > 10
        				else {m=j+1;numoftype-=tmptype;}
        				break; // end here if left + right
        			}
        			if(recordpos[m].y-recordpos[k].y>dS) {m--;break;}//SSB+
        			if(recordpos[m].w!=recordpos[k].w) numoftype++;
    				m++;
        		}
        		if(flag==0) // SSB ? 
        		{
        			complexity[2]++; // SSB+ ?
	        	 	if(numoftype!=0) results[2]++; 
	        		else results[recordpos[k].w]++;//=m-k;
        		}
        		else if(flag==2)
        		{
        			complexity[3]++; // 2SSB
	        	 	if(numoftype!=0) results[2]++;
	        		else results[recordpos[k].w]++;
        		}
	        	else
	        	{//if flag=1,m must be k+1 and from k there must be a DSB
	        		m=k;//in consitent with the calculation of chem type,
	        		numoftype=0;
	        		int numofchem=0;
	        		while(m<i)
	        		{
	        			if(recordpos[m].y-recordpos[k].y<dDSB)
	        			{
	        				if(recordpos[m].w!=recordpos[k].w) numoftype++;
	        				if(recordpos[m].w==1) numofchem++;
	        				m++;
	        			}
	        			else
	        				break;
	        		}
	        		if(numofchem==1) results[6]++;
	        		else if(numoftype!=0) results[5]++;
	        		else results[3+recordpos[k].w]++;

	        		if(m-k==2) complexity[4]++; // DSB
	        		else complexity[5]++; // DSB+
	        		cur_dsb++;
	        	}
	        	k=m;
        	}       	
        }
        if(cur_dsb>1) complexity[6]++; // DSB++
        if(k==i-1)//deal with the last one in a segment
        {
        	complexity[1]++;
        	results[recordpos[k].w]++;
        }
    }

    FILE* fp = fopen("./Results/finalstat.txt","a");
	int ssbs = 0, dsbs = 0;
	for (int i = 0; i < 3; i++) ssbs += results[i];
	for (int i = 3; i < 7; i++) dsbs += results[i];

	fprintf(fp, "%d %d %d %f\n", numFiles, ssbs, dsbs, dose);
	printf("%d %d %d %f\n", numFiles, ssbs, dsbs, dose);
    
	fprintf(fp, "SSBd SSbi SSbm DSBd DSBi DSBm DSBh\n");
    for(int index = 0; index < 7; index++)
    	fprintf(fp, "%d ", results[index]);
    fprintf(fp, "\n");
    fprintf(fp, "SSB 2xSSB SSB+ 2SSB DSB DSB+ DSB++\n");
    for(int index = 0; index < 7; index++)
    	fprintf(fp, "%d ", complexity[index]);
   	fprintf(fp, "\n");
	fclose(fp);//*/
}

#endif
