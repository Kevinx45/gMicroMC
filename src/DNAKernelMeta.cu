#include "hip/hip_runtime.h"
#include "DNAKernelMeta.cuh"
#include "DNAKernel.cu"
void DNAList::initDNAMeta()
{
  int totalspace = NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META;
		int *chromatinIndex = (int*)malloc(sizeof(int)*totalspace);
		int *chromatinStart = (int*)malloc(sizeof(int)*totalspace);
		int *chromatinType = (int*)malloc(sizeof(int)*totalspace);
    
		for (int k=0; k<totalspace; k++) 
		{
			chromatinIndex[k] = -1;
			chromatinStart[k] = -1;
			chromatinType[k] = -1;
		}
    
		int totalspace_sub = NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META * 12;
		int *segmentIndex = (int*)malloc(sizeof(int)*totalspace_sub);
		int *segmentStart = (int*)malloc(sizeof(int)*totalspace_sub);
		int *segmentType = (int*)malloc(sizeof(int)*totalspace_sub);
		for (int k=0; k<totalspace_sub; k++) 
		{
			segmentIndex[k] = -1;
			segmentStart[k] = -1;
			segmentType[k] = -1;
		}


    // X-CHROMOSOMES, there are 46 of them 
    int data[6];
		std::cout << "Reading the chromosomes and types?\n";
		std::ifstream fin;
		fin.open(document["chromCoords"].GetString()); // v5 has 0,0,0 chromosome
		float fdata[3];
		// CoorBasePair *StraightChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*STRAIGHT_BP_NUM);
		float3 *chromosome = (float3*)malloc(sizeof(float3) * NUMCHROMOSOMES_META);
		int *chromosome_type = (int*)malloc(sizeof(int) * NUMCHROMOSOMES_META);
		float ttype;
		for (int i = 0; fin >> fdata[0] >> fdata[1] >> fdata[2] >> ttype; i++) { // 46 x-chromosomes
			chromosome[i].x = fdata[0];
			chromosome[i].y = fdata[1];
			chromosome[i].z = fdata[2];
			chromosome_type[i] = ttype;
			if (i < 5) printf("%f %f %f %d\n", fdata[0], fdata[1], fdata[2], chromosome_type[i]);
		}
		fin.close();


	
		// long lSize;
		// FILE* pFile=fopen("./table/WholeNucleoChromosomesTable.bin","rb");
		// fseek (pFile , 0 , SEEK_END);
	    // lSize = ftell (pFile);
	  	// rewind (pFile);
	  	// for (int i=0; i<lSize/(6*sizeof(int)); i++)
		// {
		//     fread(data,sizeof(int),6, pFile);
		//     //if(i<5) printf("%d %d %d %d %d %d\n", data[0], data[1], data[2], data[3], data[4], data[5]);
		// 	index = data[0] + data[1] * NUCLEUS_DIM + data[2] * NUCLEUS_DIM * NUCLEUS_DIM;
		// 	chromatinIndex[index] = data[3];
		// 	chromatinStart[index] = data[4];
		// 	chromatinType[index] = data[5];
		// }
		// fclose(pFile);
		
		
		CUDA_CALL(hipMalloc((void**)&dev_chromosome, NUMCHROMOSOMES_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_chromosome, chromosome, NUMCHROMOSOMES_META * sizeof(float3), hipMemcpyHostToDevice));

		CUDA_CALL(hipMalloc((void**)&dev_chromosome_type, NUMCHROMOSOMES_META * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromosome_type, chromosome_type, NUMCHROMOSOMES_META * sizeof(int), hipMemcpyHostToDevice));
		
		std::cout << "Time to read voxelized coordinates \n";
		//ifstream fin;
		// I need to figure out how to get extra coordinates
		// maybe I can store in the chromatin index as I'm not using it anyway
		fin.open(document["voxelizedCoords"].GetString());
		// ./Results/voxelized_coordinates_b_v4_connected.txt
		for (int i=0; fin >> data[0] >> data[1] >> data[2] >> data[3] >> data[4] >> data[5]; i++)
		{
			//fread(data,sizeof(int),6, pFile);
			if(i<5) printf("%d %d %d %d %d %d\n", data[0], data[1], data[2], data[3], data[4], data[5]);
			// first 3 are indicies
			if (data[3] == 0) {
				int index = data[0] + data[1] * NUCLEUS_DIM_META + data[2] * NUCLEUS_DIM_META * NUCLEUS_DIM_META;
				chromatinIndex[index] = data[3]; // index of the extra nucleosome ?
				chromatinStart[index] = data[4]; // bp index 200
				chromatinType[index] = data[5]; // type
			}
			else {
				// Step 1)
				// convert to voxel id first
				int x = data[0]; // segment sub voxel ids
				int y = data[1];
				int z = data[2];
				int xx = x / 4; // center voxel id
				int yy = y / 4;
				int zz = z / 4;
				int xxx = x % 4; // subvoxel coordinates
				int yyy = y % 4;
				int zzz = z % 4;
				// Step 2) Convert using 'convention'
				// convention :: we have 3 walls with 4 subvoxels each
				// numerated clockwise
				// walls xy xz yz
				// subvoxels [(0, 1), (1, 0), (0, -1), (-1, 0)]
				std::array <int, 2> subs[4] = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
				// ids [0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11]
				int subvoxel_id = 0;
				xxx -= 2; // center subvoxel
				yyy -= 2;
				zzz -= 2;
				std::array <int, 2> nn;
				if (zzz == -2) { // this is xy plane
					subvoxel_id += 0;
					nn = {xxx, yyy};
				}
				if (yyy == -2) { // xz
					subvoxel_id += 4;
					nn = {xxx, zzz};
				}
				if (xxx == -2) { // yz plane
					subvoxel_id += 8;
					nn = {yyy, zzz};
				}
				for (int j = 0; j < 4; j++) {
					if (nn == subs[j]) {
						subvoxel_id += j;
						break ;
					}
				}
				// index = x + y * NUCLEUS_DIM + z * NUCLEUS_DIM * NUCLEUS_DIM; // current id of the voxel
				int index = xx + yy * NUCLEUS_DIM_META + zz * NUCLEUS_DIM_META * NUCLEUS_DIM_META;
				int sub_index = index * 12; // shifted index to accommodate 12 subvoxels
				sub_index += subvoxel_id;
				// [544.5, 170.5, 93.5]
				// if ((float)xx * 11 + 5.5 == 544.5 && 
				// 	(float)yy * 11 + 5.5 == 170.5 &&
				// 	(float)zz * 11 + 5.5 == 93.5) {
				// 	cout << "index :: " << index << " " << xx << " " << yy << " " << zz << endl;
				// 	cout << "ID of a segment and type = " << sub_index << " " << data[3] << " " << data[4] << " " << data[5] << endl;
				// }
				segmentIndex[sub_index] = data[3]; // future chromosome ID
				segmentStart[sub_index] = data[4]; // segment base pair start
				segmentType[sub_index] = data[5]; // type orientation
			}
		}
		fin.close();
		std::cout << "end of reading voxelized coordinates \n\n";
		CUDA_CALL(hipMalloc((void**)&dev_chromatinIndex, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinIndex, chromatinIndex, totalspace * sizeof(int), hipMemcpyHostToDevice));//DNA index
		CUDA_CALL(hipMalloc((void**)&dev_chromatinStart, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinStart, chromatinStart, totalspace * sizeof(int), hipMemcpyHostToDevice));//# of start base in the box
		CUDA_CALL(hipMalloc((void**)&dev_chromatinType, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinType, chromatinType, totalspace * sizeof(int), hipMemcpyHostToDevice));//type of the DNA in the box
	    free(chromatinIndex);
	    free(chromatinStart);
	    free(chromatinType);
		// copying all segments into CUDA
		CUDA_CALL(hipMalloc((void**)&dev_segmentIndex, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentIndex, segmentIndex, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//DNA index
		CUDA_CALL(hipMalloc((void**)&dev_segmentStart, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentStart, segmentStart, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//# of start base in the box
		CUDA_CALL(hipMalloc((void**)&dev_segmentType, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentType, segmentType, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//type of the DNA in the box
	    free(segmentIndex);
	    free(segmentStart);
	    free(segmentType);
		// end copying segments
		// Loading segment template
		CoorBasePair *SegmentChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*SEGMENT_BP_NUM_META);
		const char *segment = document["segmentChromatinMeta"].GetString();
		printf("Segment Chromatin Table: Reading %s\n", segment);
		FILE *fpSegment = fopen(segment,"r");
		float dump_float;
    	int dump;
		float bx, by, bz, rx, ry, rz, lx, ly, lz;
	    for (int i=0; i<SEGMENT_BP_NUM_META; i++)
		{
		    fscanf(fpSegment,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			SegmentChrom[i].base.x = bx;
			SegmentChrom[i].base.y = by;
			SegmentChrom[i].base.z = bz;
			SegmentChrom[i].right.x = rx;
			SegmentChrom[i].right.y = ry;
			SegmentChrom[i].right.z = rz;
			SegmentChrom[i].left.x = lx;
			SegmentChrom[i].left.y = ly;
			SegmentChrom[i].left.z = lz;
		}
		fclose(fpSegment);
		CUDA_CALL(hipMalloc((void**)&dev_segmentChrom, SEGMENT_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_segmentChrom, SegmentChrom, SEGMENT_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));
		//

		CoorBasePair *StraightChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*STRAIGHT_BP_NUM_META);
		const char *straight = document["straightChromatinMeta"].GetString();
		printf("Straight Chromatin Table: Reading %s\n", straight);
		FILE *fpStraight = fopen(straight,"r");
		// float dump_float;
    	// int dump;
		// float bx, by, bz, rx, ry, rz, lx, ly, lz;
	    for (int i=0; i<STRAIGHT_BP_NUM_META; i++)
		{
		    fscanf(fpStraight,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			//if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			StraightChrom[i].base.x = bx;
			StraightChrom[i].base.y = by;
			StraightChrom[i].base.z = bz;
			StraightChrom[i].right.x = rx;
			StraightChrom[i].right.y = ry;
			StraightChrom[i].right.z = rz;
			StraightChrom[i].left.x = lx;
			StraightChrom[i].left.y = ly;
			StraightChrom[i].left.z = lz;
		}
		fclose(fpStraight);
		CUDA_CALL(hipMalloc((void**)&dev_straightChrom, STRAIGHT_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_straightChrom, StraightChrom, STRAIGHT_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));

		CoorBasePair *BendChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*BEND_BP_NUM_META);
		const char *bend = document["bentChromatinMeta"].GetString();
		printf("Bend Chromatin Table: Reading %s\n", bend);
	  FILE *fpBend = fopen(bend,"r");
	   for (int i=0; i<BEND_BP_NUM_META; i++)
		 {
		  fscanf(fpStraight,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			//if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			BendChrom[i].base.x = bx;
			BendChrom[i].base.y = by;
			BendChrom[i].base.z = bz;
			BendChrom[i].right.x = rx;
			BendChrom[i].right.y = ry;
			BendChrom[i].right.z = rz;
			BendChrom[i].left.x = lx;
			BendChrom[i].left.y = ly;
			BendChrom[i].left.z = lz;
		 }
		fclose(fpBend);
		CUDA_CALL(hipMalloc((void**)&dev_bendChrom, BEND_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_bendChrom, BendChrom, BEND_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));
		
		float hisx, hisy, hisz;
		float3* bendHistone = (float3*)malloc(sizeof(float3)*BEND_HISTONE_NUM_META);
		const char *bent = document["bendHistone"].GetString();
		printf("Bent Histone Table: Reading %s\n", bent);
		FILE *fpBentH = fopen(bent,"r");
	    for (int i=0; i<BEND_HISTONE_NUM_META; i++)
		{
		    fscanf(fpBentH,"%f %f %f\n", &hisx, &hisy, &hisz);
		    //if(i<5) printf("%f %f %f\n", hisx, hisy, hisz);
			bendHistone[i].x = hisx;
			bendHistone[i].y = hisy;
			bendHistone[i].z = hisz;
		}
		fclose(fpBentH);
		CUDA_CALL(hipMalloc((void**)&dev_bendHistone, BEND_HISTONE_NUM_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_bendHistone, bendHistone, BEND_HISTONE_NUM_META * sizeof(float3), hipMemcpyHostToDevice));
		
		float3 *straightHistone = (float3*)malloc(sizeof(float3)*STRAIGHT_HISTONE_NUM_META);
		const char *straiHistone = document["straightHistoneMeta"].GetString();
		printf("Straight Histone Table: Reading %s\n", straiHistone);
		FILE *fpStraiH = fopen(straiHistone,"r");
	    for (int i=0; i<STRAIGHT_HISTONE_NUM_META; i++)
		{
		    fscanf(fpStraiH,"%f %f %f\n", &hisx, &hisy, &hisz);
		    //if(i<5) printf("%f %f %f\n", hisx, hisy, hisz);
			straightHistone[i].x = hisx;
			straightHistone[i].y = hisy;
			straightHistone[i].z = hisz;
		}
		fclose(fpStraiH);
		CUDA_CALL(hipMalloc((void**)&dev_straightHistone, STRAIGHT_HISTONE_NUM_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_straightHistone, straightHistone, STRAIGHT_HISTONE_NUM_META * sizeof(float3), hipMemcpyHostToDevice));
		
		free(StraightChrom);
		free(BendChrom);	
		free(bendHistone);	
		free(straightHistone);

		//modelTableSetup(dev_chromatinIndex,dev_chromatinStart,dev_chromatinType,dev_straightChrom,dev_bendChrom,dev_straightHistone,dev_bendHistone);
		printf("DNA geometry has been loaded to GPU memory\n");	 
 std::cout<<"finished initializing metaphase\n";
}





//chemsearch starts here

void calDNAreact_radius(float* rDNA,float deltat)
{
	float k[5]={6.1,9.2,6.4,6.1,1.8};
	float tmp=sqrtf(PI*DiffusionOfOH*deltat*0.001);
	for(int i=0;i<5;i++)
	{
		rDNA[i]=k[i]/(4*PI*DiffusionOfOH)*10/6.023;//k 10^9 L/(mol*s), Diffusion 10^9 nm^2/s. t ps
		rDNA[i]=sqrtf(rDNA[i]*tmp+tmp*tmp*0.25)-tmp*0.5;
	}
	rDNA[5]=0;//histone protein absorption radius, assumed!!!
}

__device__ float3 PosToWall(int type, float3 pos, int index) 
{
	// xy xz yz
	float shiftz;
	float shifty;
	float shiftx;
	
	if (0 <= index && index < 4) { // xy
		// shift by -z
		shiftz = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shiftx = 0.0;
			shifty = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shiftx = 5.5/2.0;
			shifty = 0;
		}	
		if (index % 4 == 2) {
			shiftx = 0.0;
			shifty = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shiftx = -5.5/2.0;
			shifty = 0;
		}	
	}
	if (4 <= index && index < 8) { // xz
		// shift by -y
		shifty = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shiftx = 0.0;
			shiftz = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shiftx = 5.5/2.0;
			shiftz = 0;
		}	
		if (index % 4 == 2) {
			shiftx = 0.0;
			shiftz = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shiftx = -5.5/2.0;
			shiftz = 0;
		}	
	}
	if (8 <= index && index < 12) { // yz
		// shift by -x
		shiftx = -5.5;
		// array <int, 2> subs = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
		if (index % 4 == 0) {
			shifty = 0.0;
			shiftz = 5.5/2.0;
		}	
		if (index % 4 == 1) {
			shifty = 5.5/2.0;
			shiftz = 0;
		}	
		if (index % 4 == 2) {
			shifty = 0.0;
			shiftz = -5.5/2.0;
		}	
		if (index % 4 == 3) {
			shifty = -5.5/2.0;
			shiftz = 0;
		}	
	}
	// shift = segment center point
	// this is to shift radical position to the
	// center of the segment
	// imagine they are close to each other
	// then we need to substruct to center radical 
	// within the segment
	pos.x = pos.x - shiftx; // relative to its center ?
	pos.y = pos.y - shifty; // 
	pos.z = pos.z - shiftz;
	float xc, yc, zc; // rotate
	switch(type)
	{
		//Straight type
	case 1:////!!!!!the following needs to be revised and confirmed
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 2://-z
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;	
		break;}
	case 3://+y
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 4:
		{xc = pos.x;
		yc = pos.z;
		zc = -pos.y;
		break;}
	case 5://+x
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;
		break;}
	case 6:
		{xc = pos.z;
		yc = pos.y;
		zc = -pos.x;
		break;}
	}
	pos.x=xc;
	pos.y=yc;
	pos.z=zc;//*/
	return pos;
}

#if RANDGEO==0
__device__ float3 pos2localMeta(int type, float3 pos, int index)
{
//do the coordinate transformation, index is the linear index for the referred box
//from global XYZ to local XYZ so that we can use the position of DNA base in two basic type (Straight and Bend) 
	int i = index%NUCLEUS_DIM_META;//the x,y,z index of the box
	int j = floorf((index%(NUCLEUS_DIM_META*NUCLEUS_DIM_META))/NUCLEUS_DIM_META);
	int k = floorf(index/NUCLEUS_DIM/NUCLEUS_DIM);
	//printf("relative to type %d %d %d %d\n", type, x,y,z);
	// this pos is the electron position which we push into the voxel
	// say N = 67
	// x is the box index [0, 67] ... 
	// here the center of the cylinder is at zero that means it can have negative coordinates
	// but x y z can't be negative as they are just box coordantes?
	// no lol
	// box is centered around zero that's okay
	// now need to push electron into it
	// x, y, z box index that we shift into global coordinate
	// 2 * x + 1 - N = [0 N] * 2 - N -> [-N N] / 2 -> [-N / 2 ; N / 2] * UL => xvec 
	// float shiftx = (2*i + 1 - NUCLEUS_DIM)*UNITLENGTH*0.5; 
	// float shifty = (2*j + 1 - NUCLEUS_DIM)*UNITLENGTH*0.5; 
	float shiftz = (k - (NUCLEUS_DIM_Z_META / 2)) * UNITLENGTH_META + UNITLENGTH_META * 0.5; 
	float shifty = (j - (NUCLEUS_DIM_META / 2)) * UNITLENGTH_META + UNITLENGTH_META * 0.5; 
	float shiftx = (i - (NUCLEUS_DIM_META / 2)) * UNITLENGTH_META + UNITLENGTH_META * 0.5; 
	pos.x = pos.x - shiftx; //relative to its center ?
	pos.y = pos.y - shifty; // 
	// pos.z = pos.z-(2*z + 1 - NUCLEUS_DIM_Z_META)*UNITLENGTH_META*0.5;
	pos.z = pos.z - shiftz;
	//printf("local coordinate %f %f %f\n", pos.x, pos.y, pos.z);
	// if (index == 27315) {
	// 	printf("It thinks the index is :: %d %d %d\n", i, j, k);
	// 	printf("So it shifts by %f %f %f\n", shiftx, shifty, shiftz);
	// }
	float xc, yc, zc;
	switch(type)
	{
		//Straight type
	case 1:////!!!!!the following needs to be revised and confirmed
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 2://-z
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;	
		break;}
	case 3://+y
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 4:
		{xc = pos.x;
		yc = pos.z;
		zc = -pos.y;
		break;}
	case 5://+x
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;
		break;}
	case 6:
		{xc = pos.z;
		yc = pos.y;
		zc = -pos.x;
		break;}
	case 7://Bend
		{xc = pos.x;
		yc = pos.y;
		zc = pos.z;
		break;}
	case 8:
		{xc = -pos.z;//Rz(pi)Ry(pi/2) [-Ry(pi/2)] 
		yc = -pos.y;
		zc = -pos.x;
		break;}
	case 9:
		{xc = -pos.x;//Rz(pi)
		yc = -pos.y;
		zc = pos.z;
		break;}
	case 10:
		{xc = -pos.z;//Ry(-pi/2)
		yc = pos.y;
		zc = pos.x;	
		break;}
	case 11:
		{xc = -pos.x;//Ry(pi)
		yc = pos.y;
		zc = -pos.z;
		break;}
	case 12:
		{xc = pos.z;//Rz(pi)Ry(-pi/2)
		yc = -pos.y;
		zc = pos.x;
		break;}
	case 13:
		{xc = pos.x;//Rx(pi)
		yc = -pos.y;
		zc = -pos.z;
		break;}
	case 14:
		{xc = pos.z;//Ry(pi/2)
		yc = pos.y;
		zc = -pos.x;
		break;}
	case 15:
		{xc = pos.y;//Rz(-pi/2)
		yc = -pos.x;
		zc = pos.z;
		break;}
	case 16:
		{xc = -pos.z;//Ry(-pi/2)Rz(pi/2) +
		yc = pos.x;
		zc = -pos.y;
		break;}
	case 17:
		{xc = -pos.y;//Rz(pi/2)
		yc = pos.x;
		zc = pos.z;
		break;}
	case 18:
		{xc = -pos.z;//Rz(-pi/2)Rx(pi/2)
		yc = -pos.x;
		zc = pos.y;
		break;}
	case 19:
		{xc = pos.y;//Rz(-pi/2)Ry(pi)
		yc = pos.x;
		zc = -pos.z;
		break;}
	case 20:
		{xc = pos.z;//Rz(-pi/2)Rx(-pi/2)
		yc = -pos.x;
		zc = pos.y;
		break;}
	case 21:
		{xc = -pos.y;//Rz(pi/2)Ry(pi)
		yc = -pos.x;
		zc = -pos.z;
		break;}
	case 22:
		{xc = pos.z;//Rz(pi/2)Rx(pi/2) ??
		yc = pos.x;
		zc = pos.y;
		// -y -z +x
		break;}
	case 23:
		{xc = pos.x;//Rx(pi/2)
		yc = -pos.z;
		zc = pos.y;
		break;}
	case 24:
		{xc = -pos.y;//Rz(pi/2)Ry(pi/2)
		yc = pos.z;
		zc = -pos.x;
		break;}
	case 25:
		{xc = -pos.x;//Rx(pi/2)Ry(pi) ??
		yc = pos.z;
		zc = pos.y;
		// xzz
		break;}
	case 26:
		{xc = -pos.y;//Rx(pi/2)Rz(pi/2)
		yc = -pos.z;
		zc = pos.x;
		break;}
	case 27:
		{
		xc = pos.x;//Rx(-pi/2)
		yc =pos.z;
		zc = -pos.y;	
		break;}
	case 28:
		{xc = pos.y;//Rx(pi/2)Rz(-pi/2)
		yc = -pos.z;
		zc = -pos.x;
		// -z -x y
		break;}
	case 29:
		{xc = -pos.x;//Rx(-pi/2)Ry(pi) ?
		yc = -pos.z;
		zc = -pos.y;
		break;}
	case 30:
		{xc = pos.y;//Rz(-pi/2)Ry(-pi/2)
		yc = pos.z;
		zc = pos.x;
		break;}
	default:
	    {printf("wrong type %d\n", type);  // for test
		break;}
	}
	pos.x=xc;
	pos.y=yc;
	pos.z=zc;//*/
	return pos;
}

__device__ float dist_function_sqr(float3 &a, float3 &b, float &height_up, float &height_down) {
	// two coordinates a and b
	// a = event
	// b = X-chromosome
	// if dist in x direction is radius within the dist then ok
	// if dist in y direction is diameter + 50 distance then ok
	// if dist in z direction is event - chrom (if event is lower a-b < 0 height down)
	// height down is negative
	return (abs(a.x - b.x) <= CYLINDERRADIUS_META) 
		&& (abs(a.y - b.y) <= 50 + CYLINDERRADIUS_META * 2)
		&& (height_down <= a.z - b.z) 
		&& (a.z - b.z <= height_up);
}
__device__ bool withinCylinder(float3 &a, float3 &cylinder) {
	// check height
	// a = 7204.624023 -229.854507 14569.363281 :: 7171.703613 -365.000000 14413.875977
	if ((a.z < (cylinder.z - CYLINDERHEIGHT_META / 2)) || 
	    ((cylinder.z + CYLINDERHEIGHT_META / 2) < a.z)) {
		return 0;
	}
	// check radial distance
	if ((a.x - cylinder.x) * (a.x - cylinder.x) + 
		(a.y - cylinder.y) * (a.y - cylinder.y) > 
		CYLINDERRADIUS_META * CYLINDERRADIUS_META) {
		return 0;
	}
	return 1;
}
__global__ void chemSearch(
	int num, 
	Edeposit* d_edrop, 
	int* dev_chromatinIndex,
	int* dev_chromatinStart,
	int* dev_chromatinType, 
	CoorBasePair* dev_straightChrom, 
	CoorBasePair* dev_segmentChrom,
	CoorBasePair* dev_bendChrom,
	float3* dev_straightHistone,
	float3* dev_bendHistone, 
	combinePhysics* d_recorde,
	float3 *dev_chromosome, 
	int *dev_chromosome_type,
	int *dev_segmentIndex, 
	int *dev_segmentStart, 
	int *dev_segmentType)
{
	int id = blockIdx.x*blockDim.x+ threadIdx.x;
	hiprandState localState = cuseed[id%MAXNUMPAR2];
	float3 newpos, pos_cur_target;
	int3 index;
	CoorBasePair* chrom;
	float3 *histone;
	int chromNum, histoneNum,flag=0;
	while(id<num)
	{
		d_recorde[id].site.x=-1;//initialize
		d_recorde[id].site.y=-1;
		d_recorde[id].site.z=-1;
		d_recorde[id].site.w=-1;		
		d_recorde[id].prob1 = 1; //hiprand_uniform(&localState); // 1
		// d_recorde[id].prob1=1;
		d_recorde[id].prob2 = 0.0; // 0.6 ? 
		// threshold for prob2 ?
		
		pos_cur_target=d_edrop[id].position; // electron position / event position
		
		// if (id < 10) {
		// 	printf("Current e position :: %f %f %f\n",
		// 		pos_cur_target.x, pos_cur_target.y, pos_cur_target.z
		// 	);
		// }
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// ***********************************************************
		// here we need to modify cur position based on dev_chromosome and dev_chromosome_type
		// Step 0) skip events too far from the y=0 plane
		// if (abs(pos_cur_target.y) > 50 + CYLINDERRADIUS * 2) {
		// 	// y position is too far from the center
		// 	id+=blockDim.x*gridDim.x;
		// 	continue ;
		// }

		// Step 1) Find nearest chromosome :)
		int found_nearest_chromosome = 0;
		int id_chromosome = -1;
		for (int i = NUMCHROMOSOMES_META - 1; i >= 0 ; i--) { // 46
			float height_up = (dev_chromosome_type[i] / 2) * CYLINDERHEIGHT_META + CYLINDERHEIGHT_META / 2;
			float height_down = -(((dev_chromosome_type[i] - 1) / 2) * CYLINDERHEIGHT_META + CYLINDERHEIGHT_META / 2);
			// height_down is negative
			// type / 2 + 1 ~ 10 / 2 + 1 = 5
			if (dist_function_sqr(pos_cur_target, dev_chromosome[i], height_up, height_down)) {
				// FOUND NEAREST CHROMOSOME!
				// Step 1.1) Mark
				found_nearest_chromosome = 1;
				id_chromosome = i;
				break;
			}
		}
		// if (id < 10) {
		// 	printf("ID chromosome: Rad ID = %d Chrom ID = %d %d\n", id, id_chromosome, found_nearest_chromosome);
		// }

		if (found_nearest_chromosome == 0) {
			// Step 1.2) if for this radical we did not find
			// anything nearby, then continue to the next :) 
			id+=blockDim.x*gridDim.x;
			continue ;
		}
		
		// if (id < 10) {
		// 	printf("ID chromosome %d %d\n", id, id_chromosome);
		// }
		// if we are here means we found chromosome
		// Step 1.3) Find nearest cylinder!
		int ttype = dev_chromosome_type[id_chromosome];
		int upper_part = ttype / 2;  // typy 4 :: 4 / 2 = 2 || type 5 :: 5 / 2 = 2
		int lower_part = (ttype - 1) / 2; // type 4 :: 4 / 2 - 1 = 1 || type 5 :: 5 / 2 - 1 = 1
		float3 nearest = dev_chromosome[id_chromosome];
		// if (id < 10) {
		// 	printf("middle :: Rad %f %f %f :: Nearest %f %f %f\n", pos_cur_target.x, pos_cur_target.y, pos_cur_target.z, 
		// 	nearest.x, nearest.y, nearest.z);
		// }
		int found_cylinder = 0;
		int id_cylinder = -1;
		// check lower and upper parts, cylinders
		// now we need to redo cylinder ID, to be in range [1 10] instead of [0 9]
		for (int idy = 0; idy < upper_part; idy++) {
			// Step 1.4) Check left and right cylinders 
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - 50 - CYLINDERRADIUS_META; 
			left_shift.z = nearest.z + CYLINDERHEIGHT_META * (idy + 1);
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + 50 + CYLINDERRADIUS_META; 
			right_shift.z = nearest.z + CYLINDERHEIGHT_META * (idy + 1);
			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome;
				id_cylinder = idy + lower_part + 1;
				break;
			}
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES_META; // right side chromosome
				id_cylinder = idy + lower_part + 1;
				break;
			}
		}
		// 
		if (!found_cylinder) {
			for (int idy = 0; idy < lower_part; idy++) {
				// Step 1.4) Check left and right cylinders 
				float3 left_shift;
				left_shift.x = nearest.x + 0.0;
				left_shift.y = nearest.y - 50 - CYLINDERRADIUS_META; 
				left_shift.z = nearest.z - CYLINDERHEIGHT_META * (idy + 1); // 
				float3 right_shift;
				right_shift.x = nearest.x + 0.0;
				right_shift.y = nearest.y + 50 + CYLINDERRADIUS_META; 
				right_shift.z = nearest.z - CYLINDERHEIGHT_META * (idy + 1);
				if (withinCylinder(pos_cur_target, left_shift)) {
					pos_cur_target.x -= left_shift.x;
					pos_cur_target.y -= left_shift.y;
					pos_cur_target.z -= left_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome;
					id_cylinder = lower_part - 1 - idy;
					break;
				}
				if (withinCylinder(pos_cur_target, right_shift)) {
					pos_cur_target.x -= right_shift.x;
					pos_cur_target.y -= right_shift.y;
					pos_cur_target.z -= right_shift.z;
					found_cylinder = 1;
					id_chromosome = id_chromosome + NUMCHROMOSOMES_META; // right side chromosome
					id_cylinder = lower_part - 1 - idy;
					break;
				}		
			}
		}
		// Step 1.5) check middle part
		if (!found_cylinder) {
			// printf("Middle check\n");
			float3 left_shift;
			left_shift.x = nearest.x + 0.0;
			left_shift.y = nearest.y - CYLINDERRADIUS_META; 
			left_shift.z = nearest.z + 0.0;
			float3 right_shift;
			right_shift.x = nearest.x + 0.0;
			right_shift.y = nearest.y + CYLINDERRADIUS_META; 
			right_shift.z = nearest.z + 0.0;

			if (withinCylinder(pos_cur_target, left_shift)) {
				pos_cur_target.x -= left_shift.x;
				pos_cur_target.y -= left_shift.y;
				pos_cur_target.z -= left_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome; // left side chromosome
				id_cylinder = lower_part;
			}
			else
			if (withinCylinder(pos_cur_target, right_shift)) {
				pos_cur_target.x -= right_shift.x;
				pos_cur_target.y -= right_shift.y;
				pos_cur_target.z -= right_shift.z;
				found_cylinder = 1;
				id_chromosome = id_chromosome + NUMCHROMOSOMES_META; // right side chromosome
				id_cylinder = lower_part;
			}
		}
		// if (id < 10) {
		// 	printf("Chromosome id type :: %d %d\n", dev_chromosome_type[id_chromosome], found_cylinder);
		// }	

		if (!found_cylinder) {
			id+=blockDim.x*gridDim.x;
			continue ;
		}
		// cylinder was found and shifted appropiately
		// continue as usual
		// END OF STEP 1
		// *******************************************
		// *******************************************
		// *******************************************
		// *******************************************
		
		// printf("The shifted e position :: %f %f %f\n", 
		// 	pos_cur_target.x, pos_cur_target.y, pos_cur_target.z
		// );
		// printf("Chromosome ID :: %d and Cylinder ID :: %d\n", 
		// 	id_chromosome, id_cylinder
		// );
		// from the global coordinate (-min max) to [0 N] index coordinate
		// what we know is that z must be say 6
		index.x=floorf(pos_cur_target.x/UNITLENGTH_META) + (NUCLEUS_DIM_META/2); // 2000 
		index.y=floorf(pos_cur_target.y/UNITLENGTH_META) + (NUCLEUS_DIM_META/2);
		index.z=floorf(pos_cur_target.z/UNITLENGTH_META) + (NUCLEUS_DIM_Z_META/2);
		
		// printf("It thinks Nucleosome index is %d %d %d\n", 
		// 	index.x, index.y, index.z
		// );

		int delta=index.x+index.y*NUCLEUS_DIM_META+index.z*NUCLEUS_DIM_META*NUCLEUS_DIM_META,minindex=-1;
		float distance[3]={100},mindis=100;
		// TO DO
		// just check 1 extra voxel nearby the wall
		flag=0;
		
		// flag changed range from 0-27 to 13-14
		for(int i=0;i<27;i++) // +6 walls
		{
			int newindex = delta+neighborindex[i];
			// if (i == 13) {
			// 	printf("ID check %d %d\n", newindex, delta);
			// }

			// flag changed Z
			if(newindex<0 || newindex > NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META-1) continue;
			int type = dev_chromatinType[newindex];
			// if (i == 13) { 
			// 	printf("Type check %d\n", type);
			// }
			if(type==-1 || type==0) continue;

			newpos = pos2localMeta(type, pos_cur_target, newindex);
			if(type<7)
			{
				// if(newpos.x<(min1-SPACETOBODER) || newpos.y<(min2-SPACETOBODER) || newpos.z<(min3-SPACETOBODER) ||newpos.x>(max1+SPACETOBODER)
				//  || newpos.y>(max2+SPACETOBODER) || newpos.z>(max3+SPACETOBODER))
				// 	continue;
				chrom=dev_straightChrom;
				chromNum=STRAIGHT_BP_NUM;
				histone=dev_straightHistone;
				histoneNum=STRAIGHT_HISTONE_NUM;
			}
			else
			{
				// if(newpos.x<(min1-SPACETOBODER) || newpos.y<(min2-SPACETOBODER) || newpos.z<(min3-SPACETOBODER) ||newpos.x>(max3+SPACETOBODER)
				//  || newpos.y>(max2+SPACETOBODER) || newpos.z>(max1+SPACETOBODER))
				// 	continue;
				chrom=dev_bendChrom;
				chromNum=BEND_BP_NUM;
				histone=dev_bendHistone;
				histoneNum=BEND_HISTONE_NUM;
			}
			// for(int j=0;j<histoneNum;j++)
			// {
			// 	mindis = caldistance(newpos, histone[j])-RHISTONE;
			// 	if(mindis < 0) flag=1;
			// }
			if(flag) break;
			for(int j=0;j<chromNum;j++) // 200 nucleosome
			{
				// can take the size of base into consideration, distance should be distance-r;
				mindis=100,minindex=-1;
				distance[0] = caldistance(newpos, chrom[j].base)-RBASE;
				distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR;
				distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR;
				// if (i == 13 && j == 99) { // lol 100th 
				// 	printf("Event within voxel %f %f %f \n and Right base pair :: %f %f %f \n",
				// 		newpos.x, newpos.y, newpos.z,
				// 		chrom[j].right.x, chrom[j].right.y, chrom[j].right.z
				// 	);
				// }
				for(int iii=0;iii<3;iii++)
				{
					if(mindis>distance[iii])
					{
						mindis=distance[iii];
						minindex=iii;
					}
				}
				// if (i == 13 && j == 99) { // lol 100th 
				// 	printf("13 and 99 :: Distances are %f %f %f\n", distance[0], distance[1], distance[2]);
				// 	printf("Min of these and id of this :: %f %d\n", mindis, minindex);
				// }
				if(mindis<0)
				{
					if(minindex>0)
					{
						// if (i == 13 && j == 99) { // lol 100th 
						// 	printf("Technically Recorded 13, 99\n");
						// }
						// printf("Ids: starting bp, inside bp, cyl = %d %d %d\n", dev_chromatinStart[newindex], j, id_cylinder);
						// GEANT4  
						d_recorde[id].site.x = id_chromosome; // 
						d_recorde[id].site.y = (dev_chromatinStart[newindex]+j) + TOTALBP_META * id_cylinder;  
						d_recorde[id].site.z = 3+minindex;
						d_recorde[id].site.w = 1; // phys or chem 0/1
					}
					flag=1;
					break;
				}
				int tmp = floorf(hiprand_uniform(&localState)/0.25);
				distance[0] = caldistance(newpos, chrom[j].base)-RBASE-d_rDNA[tmp];
				distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR- d_rDNA[4];
				distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR- d_rDNA[4];
				for(int iii=0;iii<3;iii++)
				{
					if(mindis>distance[iii])
					{
						mindis=distance[iii];
						minindex=iii;
					}
				}	
				if(mindis<0)
				{
					if(minindex>0)
					{
						// event thread id 
						d_recorde[id].site.x = id_chromosome; 
						d_recorde[id].site.y = (dev_chromatinStart[newindex]+j) + TOTALBP_META * id_cylinder;  
						// X-chromosome id //  
						d_recorde[id].site.z = 3+minindex; // left or right
						d_recorde[id].site.w = 1; // chem
					}
					flag=1;
					break;
				}
			}
			if(flag) break;
		}
		// Do all 6 walls * 4 each
		// CURRENT UPDATE 05/20/2022 **************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		// *******************************************************************
		if (flag == 0) { // still not found

			for(int i = 0; i < 24 && flag == 0; i++) // +6 walls
			{
				// printf("Checking segment number %d\n", i);
				int newdelta = delta;
				// conversions
				// the first 12 are in the current voxel so we don't need to change delta
				// 
				if (i >= 12) { // xy xz yz
					if (i < 16)  // xy +1z
						newdelta = delta + NUCLEUS_DIM_META*NUCLEUS_DIM_META;
					else if (i < 20) // xz +1y
						newdelta = delta + NUCLEUS_DIM_META;
					else // yz +1x
						newdelta = delta + 1;
				}
				int newindex = newdelta * 12 + i % 12;

				// if (i == 13) {
				// 	printf("ID check %d %d\n", newindex, delta);
				// }

				// ************flag changed Z
				// printf("New index vs total volume :: %d vs %d\n", newindex, NUCLEUS_DIM*NUCLEUS_DIM*NUCLEUS_DIM_Z * 12);
				if(newindex<0 || newindex >= NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META * 12) continue;
				
				int type = dev_segmentType[newindex];
				// if (i == 13) { 
				// 	printf("Type check %d\n", type);
				// }
				// printf("Type = %d\n", type);
				if(type==-1 || type==0) continue;

				// type is not used for pos2local because we are just getting next to this cell
				// no rotation is needed here
				float3 pos_within_voxel = pos2localMeta(1, pos_cur_target, newdelta);
				// the idea here is to shift first the position within the voxel?
				// shift relative to the voxel
				// then choose to shift next to wall center?
				// 
				newpos = PosToWall(type, pos_within_voxel, i % 12);
				// printf("Id %d and relative poistion: %0.2f %0.2f %0.2f\n", i, newpos.x, newpos.y, newpos.z);
				if(type<7)
				{
					// if(newpos.x<(min1-SPACETOBODER) || newpos.y<(min2-SPACETOBODER) || newpos.z<(min3-SPACETOBODER) ||newpos.x>(max1+SPACETOBODER)
					// || newpos.y>(max2+SPACETOBODER) || newpos.z>(max3+SPACETOBODER))
					// 	continue;
					chrom=dev_segmentChrom;
					chromNum=SEGMENT_BP_NUM_META;
				}
				else {
					// it's an error :)
				}
				if(flag) break;
				for(int j=0;j<chromNum;j++) // 17 SEGMENT
				{
					// can take the size of base into consideration, distance should be distance-r;
					mindis=100,minindex=-1;
					distance[0] = caldistance(newpos, chrom[j].base)-RBASE;
					distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR;
					distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR;
					
					for(int iii=0;iii<3;iii++)
					{
						if(mindis>distance[iii])
						{
							mindis=distance[iii];
							minindex=iii;
						}
					}
					// if (i == 13 && j == 99) { // lol 100th 
					// 	printf("13 and 99 :: Distances are %f %f %f\n", distance[0], distance[1], distance[2]);
					// 	printf("Min of these and id of this :: %f %d\n", mindis, minindex);
					// }
					if(mindis<0)
					{
						if(minindex>0)
						{
							// id is correct in the sense that it belongs to the 
							// event radical id
							// so we can record here anything
							// but what we need is the Chromosome ID to distinguish different DNA
							// base pair ID for damage calculations
							// and right or left dmg pair.
							d_recorde[id].site.x = id_chromosome; 
							d_recorde[id].site.y = (dev_segmentStart[newindex]+j) + TOTALBP_META * id_cylinder;  
							d_recorde[id].site.z = 3+minindex;
							d_recorde[id].site.w = 1; // phys or chem
						}
						flag=1; // found
						break;
					}
					int tmp = floorf(hiprand_uniform(&localState)/0.25);
					distance[0] = caldistance(newpos, chrom[j].base)-RBASE-d_rDNA[tmp];
					distance[1] = caldistance(newpos,chrom[j].left)-RSUGAR- d_rDNA[4];
					distance[2] = caldistance(newpos,chrom[j].right)-RSUGAR- d_rDNA[4];
					for(int iii=0;iii<3;iii++)
					{
						if(mindis>distance[iii])
						{
							mindis=distance[iii];
							minindex=iii;
						}
					}	
					if(mindis<0)
					{
						if(minindex>0)
						{
							d_recorde[id].site.x = id_chromosome; 
							d_recorde[id].site.y = (dev_segmentStart[newindex]+j) + TOTALBP_META * id_cylinder;  
							d_recorde[id].site.z = 3+minindex;
							d_recorde[id].site.w = 1; // phys or chem
						}
						flag=1;
						break;
					}
				}
				if(flag) break;
			}
		}
		id+=blockDim.x*gridDim.x;
	}
	cuseed[id%MAXNUMPAR2]=localState;
}
