#include "hip/hip_runtime.h"
#include "DNAKernelMeta.cuh"
__constant__  int neighborindex[27];
__constant__ float min1, min2, min3, max1, max2, max3;
__constant__  float d_rDNA[72];
void DNAList::initDNAMeta()
{
  int totalspace = NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META;
		int *chromatinIndex = (int*)malloc(sizeof(int)*totalspace);
		int *chromatinStart = (int*)malloc(sizeof(int)*totalspace);
		int *chromatinType = (int*)malloc(sizeof(int)*totalspace);
    
		for (int k=0; k<totalspace; k++) 
		{
			chromatinIndex[k] = -1;
			chromatinStart[k] = -1;
			chromatinType[k] = -1;
		}
    
		int totalspace_sub = NUCLEUS_DIM_META*NUCLEUS_DIM_META*NUCLEUS_DIM_Z_META * 12;
		int *segmentIndex = (int*)malloc(sizeof(int)*totalspace_sub);
		int *segmentStart = (int*)malloc(sizeof(int)*totalspace_sub);
		int *segmentType = (int*)malloc(sizeof(int)*totalspace_sub);
		for (int k=0; k<totalspace_sub; k++) 
		{
			segmentIndex[k] = -1;
			segmentStart[k] = -1;
			segmentType[k] = -1;
		}
    
    int* dev_chromatinIndex;
		int* dev_chromatinStart;
		int* dev_chromatinType;
		CoorBasePair* dev_straightChrom;
		CoorBasePair* dev_segmentChrom;
		CoorBasePair* dev_bendChrom;
		float3* dev_straightHistone;
		float3* dev_bendHistone;
		float3* dev_chromosome;
    int *dev_chromosome_type;
		// allocating space for the segments connecting nucleosomes
		int* dev_segmentIndex;
		int* dev_segmentStart;
		int* dev_segmentType;


    // X-CHROMOSOMES, there are 46 of them 
    int data[6];
		//cout << "Reading the chromosomes and types?\n";
		ifstream fin;
		fin.open("../tables/metadna/chromosome_coordinates_v6.txt"); // v5 has 0,0,0 chromosome
		float fdata[3];
		// CoorBasePair *StraightChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*STRAIGHT_BP_NUM);
		float3 *chromosome = (float3*)malloc(sizeof(float3) * NUMCHROMOSOMES_META);
		int *chromosome_type = (int*)malloc(sizeof(int) * NUMCHROMOSOMES_META);
		float ttype;
		for (int i = 0; fin >> fdata[0] >> fdata[1] >> fdata[2] >> ttype; i++) { // 46 x-chromosomes
			chromosome[i].x = fdata[0];
			chromosome[i].y = fdata[1];
			chromosome[i].z = fdata[2];
			chromosome_type[i] = ttype;
			if (i < 5) printf("%f %f %f %d\n", fdata[0], fdata[1], fdata[2], chromosome_type[i]);
		}
		fin.close();


	
		// long lSize;
		// FILE* pFile=fopen("./table/WholeNucleoChromosomesTable.bin","rb");
		// fseek (pFile , 0 , SEEK_END);
	    // lSize = ftell (pFile);
	  	// rewind (pFile);
	  	// for (int i=0; i<lSize/(6*sizeof(int)); i++)
		// {
		//     fread(data,sizeof(int),6, pFile);
		//     //if(i<5) printf("%d %d %d %d %d %d\n", data[0], data[1], data[2], data[3], data[4], data[5]);
		// 	index = data[0] + data[1] * NUCLEUS_DIM + data[2] * NUCLEUS_DIM * NUCLEUS_DIM;
		// 	chromatinIndex[index] = data[3];
		// 	chromatinStart[index] = data[4];
		// 	chromatinType[index] = data[5];
		// }
		// fclose(pFile);
		
		
		CUDA_CALL(hipMalloc((void**)&dev_chromosome, NUMCHROMOSOMES_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_chromosome, chromosome, NUMCHROMOSOMES_META * sizeof(float3), hipMemcpyHostToDevice));

		CUDA_CALL(hipMalloc((void**)&dev_chromosome_type, NUMCHROMOSOMES_META * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromosome_type, chromosome_type, NUMCHROMOSOMES_META * sizeof(int), hipMemcpyHostToDevice));
		
		cout << "Time to read voxelized coordinates \n";
		//ifstream fin;
		// I need to figure out how to get extra coordinates
		// maybe I can store in the chromatin index as I'm not using it anyway
		fin.open(SIDESFILE);
		// ./Results/voxelized_coordinates_b_v4_connected.txt
		for (int i=0; fin >> data[0] >> data[1] >> data[2] >> data[3] >> data[4] >> data[5]; i++)
		{
			//fread(data,sizeof(int),6, pFile);
			if(i<5) printf("%d %d %d %d %d %d\n", data[0], data[1], data[2], data[3], data[4], data[5]);
			// first 3 are indicies
			if (data[3] == 0) {
				int index = data[0] + data[1] * NUCLEUS_DIM_META + data[2] * NUCLEUS_DIM_META * NUCLEUS_DIM_META;
				chromatinIndex[index] = data[3]; // index of the extra nucleosome ?
				chromatinStart[index] = data[4]; // bp index 200
				chromatinType[index] = data[5]; // type
			}
			else {
				// Step 1)
				// convert to voxel id first
				int x = data[0]; // segment sub voxel ids
				int y = data[1];
				int z = data[2];
				int xx = x / 4; // center voxel id
				int yy = y / 4;
				int zz = z / 4;
				int xxx = x % 4; // subvoxel coordinates
				int yyy = y % 4;
				int zzz = z % 4;
				// Step 2) Convert using 'convention'
				// convention :: we have 3 walls with 4 subvoxels each
				// numerated clockwise
				// walls xy xz yz
				// subvoxels [(0, 1), (1, 0), (0, -1), (-1, 0)]
				array <int, 2> subs[4] = {{0, 1}, {1, 0}, {0, -1}, {-1, 0}};
				// ids [0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11]
				int subvoxel_id = 0;
				xxx -= 2; // center subvoxel
				yyy -= 2;
				zzz -= 2;
				array <int, 2> nn;
				if (zzz == -2) { // this is xy plane
					subvoxel_id += 0;
					nn = {xxx, yyy};
				}
				if (yyy == -2) { // xz
					subvoxel_id += 4;
					nn = {xxx, zzz};
				}
				if (xxx == -2) { // yz plane
					subvoxel_id += 8;
					nn = {yyy, zzz};
				}
				for (int j = 0; j < 4; j++) {
					if (nn == subs[j]) {
						subvoxel_id += j;
						break ;
					}
				}
				// index = x + y * NUCLEUS_DIM + z * NUCLEUS_DIM * NUCLEUS_DIM; // current id of the voxel
				int index = xx + yy * NUCLEUS_DIM_META + zz * NUCLEUS_DIM_META * NUCLEUS_DIM_META;
				int sub_index = index * 12; // shifted index to accommodate 12 subvoxels
				sub_index += subvoxel_id;
				// [544.5, 170.5, 93.5]
				// if ((float)xx * 11 + 5.5 == 544.5 && 
				// 	(float)yy * 11 + 5.5 == 170.5 &&
				// 	(float)zz * 11 + 5.5 == 93.5) {
				// 	cout << "index :: " << index << " " << xx << " " << yy << " " << zz << endl;
				// 	cout << "ID of a segment and type = " << sub_index << " " << data[3] << " " << data[4] << " " << data[5] << endl;
				// }
				segmentIndex[sub_index] = data[3]; // future chromosome ID
				segmentStart[sub_index] = data[4]; // segment base pair start
				segmentType[sub_index] = data[5]; // type orientation
			}
		}
		fin.close();
		cout << "end of reading voxelized coordinates \n\n";
		CUDA_CALL(hipMalloc((void**)&dev_chromatinIndex, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinIndex, chromatinIndex, totalspace * sizeof(int), hipMemcpyHostToDevice));//DNA index
		CUDA_CALL(hipMalloc((void**)&dev_chromatinStart, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinStart, chromatinStart, totalspace * sizeof(int), hipMemcpyHostToDevice));//# of start base in the box
		CUDA_CALL(hipMalloc((void**)&dev_chromatinType, totalspace * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_chromatinType, chromatinType, totalspace * sizeof(int), hipMemcpyHostToDevice));//type of the DNA in the box
	    free(chromatinIndex);
	    free(chromatinStart);
	    free(chromatinType);
		// copying all segments into CUDA
		CUDA_CALL(hipMalloc((void**)&dev_segmentIndex, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentIndex, segmentIndex, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//DNA index
		CUDA_CALL(hipMalloc((void**)&dev_segmentStart, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentStart, segmentStart, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//# of start base in the box
		CUDA_CALL(hipMalloc((void**)&dev_segmentType, totalspace_sub * sizeof(int)));
		CUDA_CALL(hipMemcpy(dev_segmentType, segmentType, totalspace_sub * sizeof(int), hipMemcpyHostToDevice));//type of the DNA in the box
	    free(segmentIndex);
	    free(segmentStart);
	    free(segmentType);
		// end copying segments
		// Loading segment template
		CoorBasePair *SegmentChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*SEGMENT_BP_NUM_META);
		const char *segment = "./table/NucleosomeTableSegment.txt";
		printf("Straight Chromatin Table: Reading %s\n", segment);
		FILE *fpSegment = fopen(segment,"r");
		float dump_float;
    	int dump;
		float bx, by, bz, rx, ry, rz, lx, ly, lz;
	    for (int i=0; i<SEGMENT_BP_NUM_META; i++)
		{
		    fscanf(fpSegment,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			//if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			SegmentChrom[i].base.x = bx;
			SegmentChrom[i].base.y = by;
			SegmentChrom[i].base.z = bz;
			SegmentChrom[i].right.x = rx;
			SegmentChrom[i].right.y = ry;
			SegmentChrom[i].right.z = rz;
			SegmentChrom[i].left.x = lx;
			SegmentChrom[i].left.y = ly;
			SegmentChrom[i].left.z = lz;
		}
		fclose(fpSegment);
		CUDA_CALL(hipMalloc((void**)&dev_segmentChrom, SEGMENT_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_segmentChrom, SegmentChrom, SEGMENT_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));
		//

		CoorBasePair *StraightChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*STRAIGHT_BP_NUM_META);
		const char *straight = "./table/NucleosomeTable200StraightZ.txt";
		printf("Straight Chromatin Table: Reading %s\n", straight);
		FILE *fpStraight = fopen(straight,"r");
		// float dump_float;
    	// int dump;
		// float bx, by, bz, rx, ry, rz, lx, ly, lz;
	    for (int i=0; i<STRAIGHT_BP_NUM_META; i++)
		{
		    fscanf(fpStraight,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			//if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			StraightChrom[i].base.x = bx;
			StraightChrom[i].base.y = by;
			StraightChrom[i].base.z = bz;
			StraightChrom[i].right.x = rx;
			StraightChrom[i].right.y = ry;
			StraightChrom[i].right.z = rz;
			StraightChrom[i].left.x = lx;
			StraightChrom[i].left.y = ly;
			StraightChrom[i].left.z = lz;
		}
		fclose(fpStraight);
		CUDA_CALL(hipMalloc((void**)&dev_straightChrom, STRAIGHT_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_straightChrom, StraightChrom, STRAIGHT_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));

		CoorBasePair *BendChrom = (CoorBasePair*)malloc(sizeof(CoorBasePair)*BEND_BP_NUM_META);
		const char *bend = "./table/NucleosomeTable200SideZ.txt";
		printf("Bend Chromatin Table: Reading %s\n", bend);
		FILE *fpBend = fopen(bend,"r");
	    for (int i=0; i<BEND_BP_NUM; i++)
		{
		    fscanf(fpStraight,"%f %f %f %f %f %f %f %f %f %f\n", &dump_float, &bx, &by, &bz, &rx, &ry, &rz, &lx, &ly, &lz);
			dump = dump_float;
			//if(i<5) printf("%d %f %f %f %f %f %f %f %f %f\n", dump, bx, by, bz, rx, ry, rz, lx, ly, lz);
			BendChrom[i].base.x = bx;
			BendChrom[i].base.y = by;
			BendChrom[i].base.z = bz;
			BendChrom[i].right.x = rx;
			BendChrom[i].right.y = ry;
			BendChrom[i].right.z = rz;
			BendChrom[i].left.x = lx;
			BendChrom[i].left.y = ly;
			BendChrom[i].left.z = lz;
		}
		fclose(fpBend);
		CUDA_CALL(hipMalloc((void**)&dev_bendChrom, BEND_BP_NUM_META * sizeof(CoorBasePair)));
		CUDA_CALL(hipMemcpy(dev_bendChrom, BendChrom, BEND_BP_NUM_META * sizeof(CoorBasePair), hipMemcpyHostToDevice));
		
		float hisx, hisy, hisz;
		float3* bendHistone = (float3*)malloc(sizeof(float3)*BEND_HISTONE_NUM_META);
		const char *bent = "./table/BentHistonesTable1.txt";
		printf("Bent Histone Table: Reading %s\n", bent);
		FILE *fpBentH = fopen(bent,"r");
	    for (int i=0; i<BEND_HISTONE_NUM_META; i++)
		{
		    fscanf(fpBentH,"%f %f %f\n", &hisx, &hisy, &hisz);
		    //if(i<5) printf("%f %f %f\n", hisx, hisy, hisz);
			bendHistone[i].x = hisx;
			bendHistone[i].y = hisy;
			bendHistone[i].z = hisz;
		}
		fclose(fpBentH);
		CUDA_CALL(hipMalloc((void**)&dev_bendHistone, BEND_HISTONE_NUM_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_bendHistone, bendHistone, BEND_HISTONE_NUM_META * sizeof(float3), hipMemcpyHostToDevice));
		
		float3 *straightHistone = (float3*)malloc(sizeof(float3)*STRAIGHT_HISTONE_NUM_META);
		const char *straiHistone = "./table/StraightHistonesTable1.txt";
		printf("Straight Histone Table: Reading %s\n", straiHistone);
		FILE *fpStraiH = fopen(straiHistone,"r");
	    for (int i=0; i<STRAIGHT_HISTONE_NUM_META; i++)
		{
		    fscanf(fpStraiH,"%f %f %f\n", &hisx, &hisy, &hisz);
		    //if(i<5) printf("%f %f %f\n", hisx, hisy, hisz);
			straightHistone[i].x = hisx;
			straightHistone[i].y = hisy;
			straightHistone[i].z = hisz;
		}
		fclose(fpStraiH);
		CUDA_CALL(hipMalloc((void**)&dev_straightHistone, STRAIGHT_HISTONE_NUM_META * sizeof(float3)));
		CUDA_CALL(hipMemcpy(dev_straightHistone, straightHistone, STRAIGHT_HISTONE_NUM_META * sizeof(float3), hipMemcpyHostToDevice));
		
		free(StraightChrom);
		free(BendChrom);	
		free(bendHistone);	
		free(straightHistone);

		//modelTableSetup(dev_chromatinIndex,dev_chromatinStart,dev_chromatinType,dev_straightChrom,dev_bendChrom,dev_straightHistone,dev_bendHistone);
		printf("DNA geometry has been loaded to GPU memory\n");	 
cout<<"finished initializing metaphase"<<endl;
}
