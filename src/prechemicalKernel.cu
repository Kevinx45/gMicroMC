#include "hip/hip_runtime.h"
#include "prechemicalKernel.cuh"
#include "prechemical.h"

float *d_posx, *d_posy, *d_posz; // the GPU variables to store the positions of the particles (a larger memory is required to include the product of prechemical stage) 
float *d_ene_paren, *d_ttime; // initial energies of the initial particles
int *d_ptype, *d_index; // the species type of the particles (255 for empty entries or produced H2O)	
int *d_num_prod_bran, *d_prodtype_bran;
float *d_para_replace_bran;
int *d_num_bran_paren, *d_brantype_paren;
float *d_branratio_paren;
float *d_rms_therm_elec;

__device__ __constant__	int d_num_total; 
__device__ __constant__	int d_nbrantype;
__device__ __constant__	int d_max_prod_bran; 
__device__ __constant__	int d_num_replace_bran;


__device__ __constant__	int d_nparentype;
__device__ __constant__	int d_max_bran_paren; 


__device__ __constant__	int d_num_rms_para;
	

__global__ void physiochemical_decay(float *d_posx, // x position of the particles (input and output)
                                    float *d_posy,
									float *d_posz,
									int *d_ptype, // species type for products of prechemical stage, 255 for empty or produced water (output)
									int *d_num_bran_paren,
									float *d_ratio_bran_paren,
									int *d_brantype_paren,
									int *d_num_prod_bran,
									float *d_ene_paren,
									float *d_rms_therm_elec,
									float *d_para_replace_bran,
									int *d_prodtype_bran
									)
{											 
    const int tid = blockIdx.x*blockDim.x+ threadIdx.x;
	const int pid = tid;
	
	if(tid < d_num_total)
	{  
	    hiprandState localState = cuseed[pid];
		float radnum= hiprand_uniform(&localState);
		float pro=0.0f;
	    int parentype = d_ptype[tid];
	    if (tid<10)
			{
				printf("test 2: thread id=%d, d_num_total=%d, d_posx=%f, parentype=%d\n",tid, d_num_total,d_posx[tid],parentype);
			}
	    int numbran=d_num_bran_paren[parentype];
	    int brantype;
		int numprod;

		for (int i=0;i<numbran;i++)
		{
			pro+=d_ratio_bran_paren[d_max_bran_paren*parentype+i];

			if (radnum<pro)
			{			
	    		brantype=d_brantype_paren[d_max_bran_paren*parentype+i]; // branch type for that parent molecule
	    		numprod=d_num_prod_bran[brantype]; // number of product for that branch
	    		if (numprod==0)
	    		{
	    			d_ptype[tid]=255; // H20
	    		}
	    		else
	    		{
	    			// below to sample the displacement of the products
	    			float rms[2];
	    			float r[2];
	    			for (int j=0;j<2;j++)
	    			{
	    				rms[j]=0;
	    				r[j]=0;
	    			}
	    			float dir[6];
	    			for (int j=0;j<6;j++) dir[j]=0;
	    			float ene=d_ene_paren[tid];
	    			int flag=0;
	    			if (brantype==6) // parent type 5, branch type 6, hydrated electron, note: this is input file dependent (branchInfo_prechem.txt)
	    			{
	    				float rms0=0.0;
	    				for (int j=0;j<d_num_rms_para;j++)
	    				{
	    					rms0+=d_rms_therm_elec[j]*pow(ene,d_num_rms_para-j-1);
	    				}
	    				rms[0]=rms0;
	    				flag=1;	    			
	    			}
	    			else 
	    			{
	    				for (int j=0;j<2;j++)
	    				{
	    					rms[j]=d_para_replace_bran[brantype*d_num_replace_bran+j];
	    				}
	    				
	    			}
	    			float tempr, nx, ny, nz;
	    			for (int j=0;j<2;j++)
	    			{
	    				if (rms[j]!=0)
	    				{		
	    					get_distance(&localState, rms[j], &tempr,ene,flag);
	    					r[j]=tempr;
	    					get_direction(&localState, &nx, &ny, &nz);
	    					dir[3*j]=nx;
	    					dir[3*j+1]=ny;
	    					dir[3*j+2]=nz;
	    				}	    				
	    			}

					float tempx=d_posx[tid];
					float tempy=d_posy[tid];
					float tempz=d_posz[tid];
	    			float randnum = hiprand_uniform(&localState);
	    			if ((brantype==1 || brantype==3) &&randnum<0.5) // switch the two product positions
	    			{
	    				d_posx[tid]=tempx+d_para_replace_bran[d_num_replace_bran*brantype+4]*r[0]*dir[0]+d_para_replace_bran[d_num_replace_bran*brantype+5]*r[1]*dir[3];
	    				d_posy[tid]=tempy+d_para_replace_bran[d_num_replace_bran*brantype+4]*r[0]*dir[1]+d_para_replace_bran[d_num_replace_bran*brantype+5]*r[1]*dir[4];
	    				d_posz[tid]=tempz+d_para_replace_bran[d_num_replace_bran*brantype+4]*r[0]*dir[2]+d_para_replace_bran[d_num_replace_bran*brantype+5]*r[1]*dir[5];
	    				d_posx[tid+d_num_total]=tempx+d_para_replace_bran[d_num_replace_bran*brantype+2]*r[0]*dir[0]+d_para_replace_bran[d_num_replace_bran*brantype+3]*r[1]*dir[3];
	    				d_posy[tid+d_num_total]=tempy+d_para_replace_bran[d_num_replace_bran*brantype+2]*r[0]*dir[1]+d_para_replace_bran[d_num_replace_bran*brantype+3]*r[1]*dir[4];
	   					d_posz[tid+d_num_total]=tempz+d_para_replace_bran[d_num_replace_bran*brantype+2]*r[0]*dir[2]+d_para_replace_bran[d_num_replace_bran*brantype+3]*r[1]*dir[5]; 
	   					if (brantype==3)
	   					{
	   						get_electron_distance(&localState, &tempr);
    						get_direction(&localState, &nx, &ny, &nz);	    						
    						d_posx[tid+d_num_total*2]=tempx+tempr*nx;
	    					d_posy[tid+d_num_total*2]=tempy+tempr*ny;
	    					d_posz[tid+d_num_total*2]=tempz+tempr*nz; 
	    				}
	    				for (int j=0;j<numprod;j++) d_ptype[tid+d_num_total*j]=d_prodtype_bran[d_max_prod_bran*brantype+j]; // product type
	    				if (tid<100)
						{
							printf("test 5: thread id=%d, brantype=%d, parentype=%d, randnum=%f, r1=%f, r2=%f, tempr=%f, x1=%f, x2=%f, x3=%f\n",tid,brantype,parentype, randnum, r[0],d_para_replace_bran[d_num_replace_bran*brantype+4],d_para_replace_bran[d_num_replace_bran*brantype+5],d_posx[tid],d_posx[tid+d_num_total],d_posx[tid+d_num_total*2]);
						}
	    			}	    			
	    			else
	    			{
	    				for (int j=0;j<numprod;j++)
	    				{
	    					d_ptype[tid+d_num_total*j]=d_prodtype_bran[d_max_prod_bran*brantype+j]; // product type
		    				d_posx[tid+d_num_total*j]=tempx+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)]*r[0]*dir[0]+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)+1]*r[1]*dir[3];
	    					d_posy[tid+d_num_total*j]=tempy+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)]*r[0]*dir[1]+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)+1]*r[1]*dir[4];
	    					d_posz[tid+d_num_total*j]=tempz+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)]*r[0]*dir[2]+d_para_replace_bran[d_num_replace_bran*brantype+2*(j+1)+1]*r[1]*dir[5];    					
	    				}
	    			}
	    		}
	    		break;
			}
		}  

        cuseed[pid] = localState;		
    }
}

__device__ void get_distance(hiprandState *localState_pt, float rms, float *r,float ene,int flag)
{
    float sigma=rms/sqrt(3.0); //https://doi.org/10.1016/j.ejmp.2015.10.087
    float pro_max;
    float r_max;
    if (flag==1 && ene<2) // electron < 2eV following f(r)=r^2/(2*sigma^3)*exp(-r/sigma)
    {
    	pro_max=2.0/sigma*exp(-2.0); // can be computed from the f(r)'=0
    	r_max=8.0*sigma;
    }
	else
	{
		pro_max=sqrt(2.0/PI)*2.0/sigma*exp(-1.0); //f(r)=sqrt(2/pi)*r^2/(sigma^3)*exp(-r^2/(2*sigma^2))
		r_max=3.75*sigma;
	}
	float rsample=hiprand_uniform(localState_pt)*r_max;
	float pro=1.0;

	float pror=0.0;

	while(pro>pror)
	{
		rsample=hiprand_uniform(localState_pt)*r_max;
		if (flag==1 && ene<2) // electron < 2eV following f(r)=r^2/(2*sigma^3)*exp(-r/sigma)
    	{
    		pror=pow(rsample,2)/(2.0*pow(sigma,3))*exp(-rsample/sigma);
    	}
		else
		{
			pror=sqrt(2.0/PI)*pow(rsample,2)/pow(sigma,3)*exp(-rsample*rsample/(2*sigma*sigma));
		}
		pro=hiprand_uniform(localState_pt)*pro_max;
	}
	*r=rsample;
}
__device__ void get_electron_distance(hiprandState *localState_pt, float *r)
{
    float r_max=0.5f; //https://doi.org/10.1016/j.ejmp.2015.10.087 f(r)=4*r*exp(-2r)
    float pro_max=2.0*exp(-1.0);
	float rsample=hiprand_uniform(localState_pt)*r_max;
	float pro=1.0;
	float pror=0.0;

	while(pro>pror)
	{
		rsample=hiprand_uniform(localState_pt)*r_max;
		pror=4.0*rsample*exp(-2.0*rsample);
		pro=hiprand_uniform(localState_pt)*pro_max;
	}
	*r=rsample;
}										  

__device__ void get_direction(hiprandState *localState_pt, float *nx, float *ny, float *nz)                                                              					                                                              					
{// uniform sampling on a unit sphere
	float beta = hiprand_uniform(localState_pt)*2.0f*PI;
	float costheta = 1.0f-2.0f*hiprand_uniform(localState_pt);
	
	*nx = sqrtf(1-costheta*costheta) * __cosf(beta);
	*ny = sqrtf(1-costheta*costheta)  * __sinf(beta);
	*nz = costheta;
}												  

void PrechemList::initGPUVariables()
{
	
// initial particle info
	printf("total_initial parent particle to go through radiolysis is: %d\n",num_total_paren);

	hipMemcpyToSymbol(HIP_SYMBOL(d_num_total), &num_total_paren, sizeof(int), 0, hipMemcpyHostToDevice);
	
	// branch type and branch model info
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbrantype), &nbrantype, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_max_prod_bran), &max_prod_bran, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_num_replace_bran), &num_replace_bran, sizeof(int), 0, hipMemcpyHostToDevice);
    // parent molecule type and decay branch info for each parent
	hipMemcpyToSymbol(HIP_SYMBOL(d_nparentype), &nparentype, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_max_bran_paren), &max_bran_paren, sizeof(int), 0, hipMemcpyHostToDevice);
	
    // electron thermalization rms para
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_rms_para), &num_para_recom[1], sizeof(int), 0, hipMemcpyHostToDevice);
  
	printf("finish memory copy for prechemcial stage\n");
}

void PrechemList::run()
{
	//simulating the prechemical stage for the subexcitation electrons: thermalisation or recombination with its parent ionized water
    int nblocks = 1 + (num_total_paren - 1)/NTHREAD_PER_BLOCK_PAR;
    printf("start prechemical run with number of blocks: %d\n", nblocks);
	physiochemical_decay<<<nblocks,NTHREAD_PER_BLOCK_PAR>>>(posx_paren, posy_paren,posz_paren,type_paren,num_bran_paren,branratio_paren,brantype_paren,num_prod_bran,ene_paren,rms_therm_elec,
		para_replace_bran,prodtype_bran);
	//printf("end prechemical run with number of blocks: %d\n", nblocks);
	hipDeviceSynchronize();	
}

void PrechemList::saveResults()
{
	FILE *fp;
	
	//remove the empty entries or H2O entries from the particle data
	thrust::device_ptr<float> posx_dev_ptr;
	thrust::device_ptr<float> posy_dev_ptr;
	thrust::device_ptr<float> posz_dev_ptr;
	thrust::device_ptr<int> ptype_dev_ptr;
	thrust::device_ptr<int> index_dev_ptr;
	thrust::device_ptr<float> ttime_dev_ptr;
	
	typedef thrust::tuple<thrust::device_vector<int>::iterator, thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator, thrust::device_vector<int>::iterator, thrust::device_vector<float>::iterator> IteratorTuple;
        // define a zip iterator
	typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
	
	ZipIterator zip_begin, zip_end, zip_new_end;
	
	ptype_dev_ptr = thrust::device_pointer_cast(&type_paren[0]);		
	posx_dev_ptr = thrust::device_pointer_cast(&posx_paren[0]);	
	posy_dev_ptr = thrust::device_pointer_cast(&posy_paren[0]);	
	posz_dev_ptr = thrust::device_pointer_cast(&posz_paren[0]);	
	index_dev_ptr = thrust::device_pointer_cast(&index_paren[0]);
	ttime_dev_ptr = thrust::device_pointer_cast(&ttime_paren[0]);

	zip_begin = thrust::make_zip_iterator(thrust::make_tuple(ptype_dev_ptr, posx_dev_ptr, posy_dev_ptr, posz_dev_ptr, index_dev_ptr, ttime_dev_ptr));
	zip_end   = zip_begin + num_total_paren * 3;  		
	zip_new_end = thrust::remove_if(zip_begin, zip_end, first_element_equal_255());
	
	hipDeviceSynchronize();
	
	int	numCurPar = zip_new_end - zip_begin;
		
	printf("After removing, numCurPar = %d\n", numCurPar);
	float *output_posx = (float*)malloc(sizeof(float) * numCurPar);
    float *output_posy = (float*)malloc(sizeof(float) * numCurPar);
    float *output_posz = (float*)malloc(sizeof(float) * numCurPar);
    float *output_ttime = (float*)malloc(sizeof(float) * numCurPar);
    int *output_ptype = (int*)malloc(sizeof(float) * numCurPar);
    int *output_index = (int*)malloc(sizeof(float) * numCurPar);
    
    memcpy(output_posx , posx_paren, sizeof(float)*numCurPar);	
    memcpy(output_posy , posy_paren, sizeof(float)*numCurPar);	
    memcpy(output_posz , posz_paren, sizeof(float)*numCurPar);
    memcpy(output_ptype, type_paren, sizeof(int)*numCurPar);	
    memcpy(output_index, index_paren, sizeof(int)*numCurPar);	
    memcpy(output_ttime, ttime_paren, sizeof(int)*numCurPar);	
	
	std::string fname = document["fileForOutput"].GetString();
	fp = fopen(fname.c_str(), "wb");	
    fwrite(output_posx, sizeof(float), numCurPar, fp);
    fwrite(output_posy, sizeof(float), numCurPar, fp);
	fwrite(output_posz, sizeof(float), numCurPar, fp);
	fwrite(output_ttime, sizeof(float), numCurPar, fp);
	fwrite(output_index, sizeof(int), numCurPar, fp);
	fwrite(output_ptype, sizeof(int), numCurPar, fp);
	fclose(fp);	
	

    hipFree(num_prod_bran);
	hipFree(prodtype_bran);
	hipFree(para_replace_bran);

	hipFree(num_bran_paren);
	hipFree(brantype_paren);
	hipFree(branratio_paren);

	hipFree(pro_recom);
	hipFree(rms_therm_elec);

	hipFree(posx_paren);
	hipFree(posy_paren);
	hipFree(posz_paren);
	hipFree(ene_paren);
	hipFree(ttime_paren);
	hipFree(type_paren);
	hipFree(index_paren);

}
